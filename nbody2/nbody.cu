#include "hip/hip_runtime.h"
/*
 *
 * nbody.cu
 *
 * N-body example that illustrates gravitational simulation.
 * This is the type of computation that GPUs excel at:
 * parallelizable, with lots of FLOPS per unit of external 
 * memory bandwidth required.
 *
 * Requires: No minimum SM requirement.  If SM 3.x is not available,
 * this application quietly replaces the shuffle and fast-atomic
 * implementations with the shared memory implementation.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

// for kbhit()
#include <ch_conio.h>

#include <math.h>

#include <chCommandLine.h>
#include <chError.h>
#include <chThread.h>
#include <chTimer.h>

#include "nbody.h"
#include "kahan.h"

#include "bodybodyInteraction.cuh"
#include "bodybodyInteraction_SSE.h"

using namespace cudahandbook::threading;

inline void
randomVector( float v[3] )
{
    float lenSqr;
    do {
        v[0] = rand() / (float) RAND_MAX * 2 - 1;
        v[1] = rand() / (float) RAND_MAX * 2 - 1;
        v[2] = rand() / (float) RAND_MAX * 2 - 1;
        lenSqr = v[0]*v[0]+v[1]*v[1]+v[2]*v[2];
    } while ( lenSqr > 1.0f );
}

void
randomUnitBodies( int seed, std::vector<PosMass<float>>& pos, std::vector<VelInvMass<float>>& vel, size_t N )
{
    float r[3];
    srand( seed );
    for ( auto& p: pos ) {
        randomVector( &r[0] );
        p.x_ = r[0];
        p.y_ = r[1];
        p.z_ = r[2];
        p.mass_ = 1.0f;  // unit mass
    }
    for ( auto& v: vel ) {
        randomVector( &r[0] );
        v.dx_ = r[0];
        v.dy_ = r[1];
        v.dz_ = r[2];
        v.invMass_ = 1.0f;
    }
}

bool g_bCUDAPresent;
bool g_bSM30Present;

std::vector<PosMass<float>> g_hostAOS_PosMass;
std::vector<VelInvMass<float>> g_hostAOS_VelInvMass;
std::vector<Force3D<float>> g_hostAOS_Force;//float *g_hostAOS_Force;
float *g_hostAOS_gpuCrossCheckForce[32];

float *g_dptrAOS_PosMass;
float *g_dptrAOS_Force;

//
// threshold for soft comparisons when validating
// that forces add up to 0.
//
double g_ZeroThreshold;

bool g_bGPUTest;

// Buffer to hold the golden version of the forces, used for comparison
// Along with timing results, we report the maximum relative error with 
// respect to this array.
std::vector<Force3D<float>> g_hostAOS_Force_Golden;//float *g_hostAOS_Force_Golden;

float *g_hostSOA_Pos[3];
float *g_hostSOA_Force[3];
float *g_hostSOA_Mass;
float *g_hostSOA_InvMass;

size_t g_N;

float g_softening = 0.1f;
float g_damping = 0.995f;
float g_dt = 0.016f;

template<typename T>
static T
relError( T a, T b )
{
    if ( a == b ) return 0.0f;
    T relErr = (a-b)/b;
    // Manually take absolute value
    return (relErr<0.0f) ? -relErr : relErr;
}

template<typename T>
static T
absError( T a, T b )
{
    if ( a == b ) return 0.0f;
    return (T) fabs(a-b);
}


#include "nbody_CPU_AOS.h"

//#include "nbody_CPU_AOS_tiled.h"
//#include "nbody_CPU_SOA.h"
//#include "nbody_CPU_SIMD.h"

#ifndef NO_CUDA
#include "nbody_GPU_AOS.cuh"
//#include "nbody_GPU_AOS_const.cuh"
//#include "nbody_GPU_AOS_tiled.cuh"
//#include "nbody_GPU_AOS_tiled_const.cuh"
//#include "nbody_GPU_SOA_tiled.cuh"
//#include "nbody_GPU_Shuffle.cuh"
//#include "nbody_GPU_Atomic.cuh"
#endif

template<typename T>
void
NBodyAlgorithm<T>::integrateGravitation( T dt, T damping )
{
    for ( size_t i = 0; i < N_; i++ ) {
        float pos[3] = { posMass_[i].x_, posMass_[i].y_, posMass_[i].z_ };
        float vel[3] = { velInvMass_[i].dx_, velInvMass_[i].dy_, velInvMass_[i].dz_ };
        float invMass = velInvMass_[i].invMass_;
        float force[3] = { force_[i].ddx_, force_[i].ddy_, force_[i].ddz_ };

        // acceleration = force / mass;
        // new velocity = old velocity + acceleration * deltaTime
        vel[0] += (force[0] * invMass) * dt;
        vel[1] += (force[1] * invMass) * dt;
        vel[2] += (force[2] * invMass) * dt;

        vel[0] *= damping;
        vel[1] *= damping;
        vel[2] *= damping;

        // new position = old position + velocity * deltaTime
        pos[0] += vel[0] * dt;
        pos[1] += vel[1] * dt;
        pos[2] += vel[2] * dt;

        posMass_[i].x_ = pos[0];
        posMass_[i].y_ = pos[1];
        posMass_[i].z_ = pos[2];

        velInvMass_[i].dx_ = vel[0];
        velInvMass_[i].dy_ = vel[1];
        velInvMass_[i].dz_ = vel[2];
    }
}

enum nbodyAlgorithm_enum g_Algorithm;

//
// g_maxAlgorithm is used to determine when to rotate g_Algorithm back to CPU_AOS
// If CUDA is present, it is CPU_SIMD_threaded, otherwise it depends on SM version
//
// The shuffle and tiled implementations are SM >=3.0 only.
//
// The CPU and GPU algorithms must be contiguous, and the logic in main() to
// initialize this value must be modified if any new algorithms are added.
//
enum nbodyAlgorithm_enum g_maxAlgorithm;
bool g_bCrossCheck = true;
bool g_bUseSIMDForCrossCheck = true;
bool g_bNoCPU = false;
bool g_bGPUCrossCheck = false;
bool g_bGPUCrossCheckFile = false;
FILE *g_fGPUCrosscheckInput;
FILE *g_fGPUCrosscheckOutput;

template<typename T>
bool
NBodyAlgorithm<T>::Initialize( size_t N, int seed, T softening )
{
    N_ = N;
    softening_ = softening;
    force_ = std::vector<Force3D<T>>( N );
    posMass_ = std::vector<PosMass<T>>( N );
    velInvMass_ = std::vector<VelInvMass<T>>( N );
    randomUnitBodies( seed, posMass_, velInvMass_, N );
    return true;
}

//
// 
//
template<typename T>
bool
NBodyAlgorithm_GPU<T>::Initialize( size_t N, int seed, T softening )
{
    hipError_t status;
    if ( ! NBodyAlgorithm<T>::Initialize( N, seed, softening ) )
        return false;
    cuda(EventCreate( &evStart_ ) );
    cuda(EventCreate( &evStop_ ) );
    gpuForce_ = thrust::device_vector<Force3D<float>>( N );
    gpuPosMass_ = thrust::device_vector<PosMass<float>>( N );
    gpuVelInvMass_ = thrust::device_vector<VelInvMass<float>>( N );
    return true;
Error:
    return false;
}

template<typename T>
inline bool
NBodyAlgorithm_SOA<T>::Initialize( size_t N, int seed, T softening )
{
    NBodyAlgorithm<T>::Initialize( N, seed, softening );
    x_ = std::vector<T>( N );
    y_ = std::vector<T>( N );
    z_ = std::vector<T>( N );
    mass_ = std::vector<T>( N );
    ddx_ = std::vector<T>( N );
    ddy_ = std::vector<T>( N );
    ddz_ = std::vector<T>( N );
    return true;
}

template<typename T>
float
NBodyAlgorithm<T>::computeTimeStep( )
{
    T softeningSquared = softening_*softening_;
    chTimerTimestamp start, end;
    chTimerGetTime( &start );
    for ( size_t i = 0; i < N_; i++ )
    {
        Force3D<T> acc = { 0, 0, 0 };
        float myX = posMass_[i].x_;
        float myY = posMass_[i].y_;
        float myZ = posMass_[i].z_;

        for ( size_t j = 0; j < N_; j++ ) {
            if ( i==j ) continue;
            float fx, fy, fz;
            float bodyX = posMass_[j].x_;
            float bodyY = posMass_[j].y_;
            float bodyZ = posMass_[j].z_;
            float bodyMass = posMass_[j].mass_;

            bodyBodyInteraction<float>(
                &fx, &fy, &fz,
                myX, myY, myZ,
                bodyX, bodyY, bodyZ, bodyMass,
                softeningSquared );
            acc.ddx_ += fx;
            acc.ddy_ += fy;
            acc.ddz_ += fz;
        }

        force_[i].ddx_ = acc.ddx_;
        force_[i].ddy_ = acc.ddy_;
        force_[i].ddz_ = acc.ddz_;

    }
    chTimerGetTime( &end );
    return (float) chTimerElapsedTime( &start, &end ) * 1000.0f;
}

template<typename T>
float
NBodyAlgorithm_SOA<T>::computeTimeStep( )
{
    auto posMass = NBodyAlgorithm<T>::posMass();
    auto& force = NBodyAlgorithm<T>::force();
    size_t N = NBodyAlgorithm<T>::N();
    T softeningSquared = NBodyAlgorithm<T>::softening()*NBodyAlgorithm<T>::softening();
    chTimerTimestamp start, end;
    chTimerGetTime( &start );
    for ( size_t i = 0; i < N; i++ ) {
        x_[i] = posMass[i].x_;
        y_[i] = posMass[i].y_;
        z_[i] = posMass[i].z_;
        mass_[i] = posMass[i].mass_;
    }
    for ( size_t i = 0; i < N; i++ )
    {
        Force3D<T> acc = { 0, 0, 0 };
        float myX = x_[i];
        float myY = y_[i];
        float myZ = z_[i];

        for ( size_t j = 0; j < N; j++ ) {
            if ( i == j ) continue;
            float fx, fy, fz;
            float bodyX = x_[j];
            float bodyY = y_[j];
            float bodyZ = z_[j];
            float bodyMass = mass_[j];

            bodyBodyInteraction<float>(
                &fx, &fy, &fz,
                myX, myY, myZ,
                bodyX, bodyY, bodyZ, bodyMass,
                softeningSquared );
            acc.ddx_ += fx;
            acc.ddy_ += fy;
            acc.ddz_ += fz;
        }

        ddx_[i] = acc.ddx_;
        ddy_[i] = acc.ddy_;
        ddz_[i] = acc.ddz_;
    }
    for ( size_t i = 0; i < N; i++ ) {
        force[i].ddx_ = ddx_[i];
        force[i].ddy_ = ddy_[i];
        force[i].ddz_ = ddz_[i];
    }
    chTimerGetTime( &end );
    return (float) chTimerElapsedTime( &start, &end ) * 1000.0f;
}

template<typename T>
float
NBodyAlgorithm_SSE<T>::computeTimeStep( )
{
    size_t N = NBodyAlgorithm<T>::N();
    if ( 0 != N%4 )
        return 0.0f;

    auto posMass = NBodyAlgorithm<T>::posMass();
    auto& force = NBodyAlgorithm<T>::force();

    auto x = NBodyAlgorithm_SOA<T>::x();
    auto y = NBodyAlgorithm_SOA<T>::y();
    auto z = NBodyAlgorithm_SOA<T>::z();
    auto mass = NBodyAlgorithm_SOA<T>::mass();
    auto& ddx = NBodyAlgorithm_SOA<T>::ddx();
    auto& ddy = NBodyAlgorithm_SOA<T>::ddy();
    auto& ddz = NBodyAlgorithm_SOA<T>::ddz();

    T softeningSquared = NBodyAlgorithm<T>::softening()*NBodyAlgorithm<T>::softening();
    chTimerTimestamp start, end;
    chTimerGetTime( &start );
    for ( size_t i = 0; i < N; i++ ) {
        x[i] = posMass[i].x_;
        y[i] = posMass[i].y_;
        z[i] = posMass[i].z_;
        mass[i] = posMass[i].mass_;
    }


    for (int i = 0; i < N; i++)
    {
        __m128 ax = _mm_setzero_ps();
        __m128 ay = _mm_setzero_ps();
        __m128 az = _mm_setzero_ps();
        __m128 *px = (__m128 *) x.data();
        __m128 *py = (__m128 *) y.data();
        __m128 *pz = (__m128 *) z.data();
        __m128 *pmass = (__m128 *) mass.data();
        float *pddx = (float *) ddx.data();
        float *pddy = (float *) ddy.data();
        float *pddz = (float *) ddz.data();
        __m128 x0 = _mm_set_ps1( x[i] );
        __m128 y0 = _mm_set_ps1( y[i] );
        __m128 z0 = _mm_set_ps1( z[i] );
        __m128i j4 = _mm_set_epi32( 3, 2, 1, 0 );

        for ( int j = 0; j < N/4; j++ ) {
            bodyBodyInteraction(
                ax, ay, az,
                x0, y0, z0,
                px[j], py[j], pz[j], pmass[j],
                _mm_set_ps1( softeningSquared ),
                _mm_castsi128_ps( _mm_cmpeq_epi32( j4, _mm_set1_epi32( i ) ) ) );
            j4 = _mm_add_epi32( j4, _mm_set1_epi32( 4 ) );
        }

        auto horizontal_sum_ps = []( const __m128 x ) -> __m128 {
            const __m128 t = _mm_add_ps(x, _mm_movehl_ps(x, x));
            return _mm_add_ss(t, _mm_shuffle_ps(t, t, 1));
        };

        // Accumulate sum of four floats in the SSE register
        ax = horizontal_sum_ps( ax );
        ay = horizontal_sum_ps( ay );
        az = horizontal_sum_ps( az );

        _mm_store_ss( &pddx[i], ax );
        _mm_store_ss( &pddy[i], ay );
        _mm_store_ss( &pddz[i], az );
    }
    for ( size_t i = 0; i < N; i++ ) {
        force[i].ddx_ = ddx[i];
        force[i].ddy_ = ddy[i];
        force[i].ddz_ = ddz[i];
    }
    chTimerGetTime( &end );
    return (float) chTimerElapsedTime( &start, &end ) * 1000.0f;
}

template<typename T>
__global__ void
ComputeNBodyGravitation_GPU_AOS(
    Force3D<T> *force,
    PosMass<T> *posMass,
    T softeningSquared,
    size_t N )
{
    for ( int i = blockIdx.x*blockDim.x + threadIdx.x;
              i < N;
              i += blockDim.x*gridDim.x )
    {
        T acc[3] = {0};
        float4 me = ((float4 *) posMass)[i];
        T myX = me.x;
        T myY = me.y;
        T myZ = me.z;
        for ( int j = 0; j < N; j++ ) {
            float4 body = ((float4 *) posMass)[j];
            float fx, fy, fz;
            bodyBodyInteraction(
                &fx, &fy, &fz,
                myX, myY, myZ,
                body.x, body.y, body.z, body.w,
                softeningSquared);
            acc[0] += fx;
            acc[1] += fy;
            acc[2] += fz;
        }
        force[i].ddx_ = acc[0];
        force[i].ddy_ = acc[1];
        force[i].ddz_ = acc[2];
    }
}

template<typename T>
float
NBodyAlgorithm_GPU<T>::computeTimeStep( )
{
    hipError_t status;
    float ms = 0.0f;
    float softeningSquared = NBodyAlgorithm<T>::softening()*NBodyAlgorithm<T>::softening();

    cuda(Memcpy( thrust::raw_pointer_cast(gpuPosMass_.data()), NBodyAlgorithm<T>::posMass().data(), NBodyAlgorithm<T>::N()*sizeof(PosMass<float>), hipMemcpyHostToDevice ) );
    cuda(EventRecord( evStart_, NULL ) );
    ComputeNBodyGravitation_GPU_AOS<<<1024,256>>>(
        thrust::raw_pointer_cast(gpuForce_.data()),
        thrust::raw_pointer_cast(gpuPosMass_.data()),
        softeningSquared,
        NBodyAlgorithm<T>::N() );
    cuda(EventRecord( evStop_, NULL ) );
    cuda(DeviceSynchronize() );
    cuda(Memcpy( NBodyAlgorithm<T>::force().data(), thrust::raw_pointer_cast(gpuForce_.data()), NBodyAlgorithm<T>::N()*sizeof(Force3D<float>), hipMemcpyDeviceToHost ) );
    cuda(EventElapsedTime( &ms, evStart_, evStop_ ) );
Error:
    return ms;
}

template<typename T>
bool
ComputeGravitation( 
    float *ms,
    float *maxAbsError,
    NBodyAlgorithm<T> *refAlgo,
    NBodyAlgorithm<T> *gpuAlgo,
    bool bCrossCheck )
{
    //hipError_t status;
    //bool bSOA = false;

#if 0
    if ( bCrossCheck ) {
#ifdef HAVE_SIMD_THREADED
        if ( g_bUseSIMDForCrossCheck ) {
            ComputeGravitation_SIMD_threaded(
                            g_hostSOA_Force,
                            g_hostSOA_Pos,
                            g_hostSOA_Mass,
                            g_softening*g_softening,
                            g_N );
            for ( size_*t i = 0; i < g_N; i++ ) {
                g_hostAOS_Force_Golden[i].dx_ = g_hostSOA_Force[0][i];
                g_hostAOS_Force_Golden[i].dy_ = g_hostSOA_Force[1][i];
                g_hostAOS_Force_Golden[i].dz_ = g_hostSOA_Force[2][i];
            }
        }
        else {
#endif
            ComputeGravitation_AOS( 
                g_hostAOS_Force_Golden,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
#ifdef HAVE_SIMD_THREADED
        }
#endif
    }
#endif

#if 0
    // CPU->GPU copies in case we are measuring GPU performance
    if ( g_bCUDAPresent ) {
        cuda(MemcpyAsync( 
            g_dptrAOS_PosMass, 
            g_hostAOS_PosMass.data(), 
            4*g_N*sizeof(float), 
            hipMemcpyHostToDevice ) );
    }
#endif

    *maxAbsError = 0.0f;

    if ( bCrossCheck ) {
        auto gpuPosMass = gpuAlgo->posMass();
        auto refPosMass = refAlgo->posMass();
        float max = 0.0f;
        for ( size_t i = 0; i < g_N; i++ ) {
            float xerr = absError( gpuPosMass[i].x_, refPosMass[i].x_ );
            float yerr = absError( gpuPosMass[i].y_, refPosMass[i].y_ );
            float zerr = absError( gpuPosMass[i].z_, refPosMass[i].z_ );
            if ( xerr > max ) max = xerr;
            if ( yerr > max ) max = yerr;
            if ( zerr > max ) max = zerr;
        }
        *maxAbsError = max;
#if 0
        printf( "%s crosscheck against gold %s: maxAbsError = gpuForce-> %E\n", 
            gpuAlgo->getAlgoName(), refAlgo->getAlgoName(), max ); fflush( stdout );
#endif
    }

    *ms = gpuAlgo->computeTimeStep( );

#if 0
    switch ( algorithm ) {
        case CPU_AOS:
        default:
            *ms = ComputeGravitation_AOS( 
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
#if 0
        case CPU_AOS_tiled:
            *ms = ComputeGravitation_AOS_tiled( 
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
        case CPU_SOA:
            *ms = ComputeGravitation_SOA(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = true;
            break;
#ifdef HAVE_SIMD
        case CPU_SIMD:
            *ms = ComputeGravitation_SIMD(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = true;
            break;
#endif
#ifdef HAVE_SIMD_THREADED
        case CPU_SIMD_threaded:
            *ms = ComputeGravitation_SIMD_threaded(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = true;
            break;
#endif
#ifdef HAVE_SIMD_OPENMP
        case CPU_SIMD_openmp:
            *ms = ComputeGravitation_SIMD_openmp(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = true;
            break;
#endif
#ifndef NO_CUDA
        case GPU_AOS:
            *ms = ComputeGravitation_GPU_AOS( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            cuda(Memcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_AOS_tiled:
            *ms = ComputeGravitation_GPU_AOS_tiled( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            cuda(Memcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_AOS_tiled_const:
            *ms = ComputeGravitation_GPU_AOS_tiled_const( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            cuda(Memcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
#if 0
// commented out - too slow even on SM 3.0
        case GPU_Atomic:
            cuda(Memset( g_dptrAOS_Force, 0, 3*sizeof(float) ) );
            *ms = ComputeGravitation_GPU_Atomic( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            cuda(Memcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
#endif
        case GPU_Shared:
            cuda(Memset( g_dptrAOS_Force, 0, 3*g_N*sizeof(float) ) );
            *ms = ComputeGravitation_GPU_Shared( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            cuda(Memcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_Const:
            cuda(Memset( g_dptrAOS_Force, 0, 3*g_N*sizeof(float) ) );
            *ms = ComputeNBodyGravitation_GPU_AOS_const( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            cuda(Memcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_Shuffle:
            cuda(Memset( g_dptrAOS_Force, 0, 3*g_N*sizeof(float) ) );
            *ms = ComputeGravitation_GPU_Shuffle( 
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            cuda(Memcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case multiGPU_SingleCPUThread:
            memset( g_hostAOS_Force, 0, 3*g_N*sizeof(float) );
            *ms = ComputeGravitation_multiGPU_singlethread( 
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
        case multiGPU_MultiCPUThread:
            memset( g_hostAOS_Force, 0, 3*g_N*sizeof(float) );
            *ms = ComputeGravitation_multiGPU_threaded( 
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
#endif
        default:
            fprintf(stderr, "Unrecognized algorithm index: %d\n", algorithm);
            abort();
            break;
#endif
    }

    if ( g_bGPUCrossCheck ) {
        int cDisagreements = 0;
        for ( int i = 0; i < g_numGPUs; i++ ) {
            for ( int j = 1; j < g_numGPUs; j++ ) {
                if ( memcmp( g_hostAOS_gpuCrossCheckForce[i], 
                             g_hostAOS_gpuCrossCheckForce[j], 
                             3*g_N*sizeof(float) ) ) {
                    fprintf( stderr, "GPU %d and GPU %d disagreed\n", i, j );
                    cDisagreements += 1;
                }
            }
        }
        if ( cDisagreements ) {
            goto Error;
        }
    }


    // SOA -> AOS
    if ( bSOA ) {
        for ( size_t i = 0; i < g_N; i++ ) {
            g_hostAOS_Force[i].ddx_ = g_hostSOA_Force[0][i];
            g_hostAOS_Force[i].ddy_ = g_hostSOA_Force[1][i]; 
            g_hostAOS_Force[i].ddz_ = g_hostSOA_Force[2][i];
        }
    }
#endif

    *maxAbsError = 0.0f;

    if ( bCrossCheck ) {
        float msGPU = refAlgo->computeTimeStep( );
        auto gpuForce = gpuAlgo->force();
        auto refForce = refAlgo->force();
        float max = 0.0f;
        for ( size_t i = 0; i < g_N; i++ ) {
            float xerr = absError( gpuForce[i].ddx_, refForce[i].ddx_ );
            float yerr = absError( gpuForce[i].ddy_, refForce[i].ddy_ );
            float zerr = absError( gpuForce[i].ddz_, refForce[i].ddz_ );
#if 0
            if ( xerr >= 1.0f || yerr >= 1.0f || zerr >= 1.0 ) {
                asm("int $3");
            }
#endif
            if ( xerr > max ) max = xerr;
            if ( yerr > max ) max = yerr;
            if ( zerr > max ) max = zerr;
        }
        *maxAbsError = max;
        //printf( "%s crosscheck against gold %s: maxAbsError = gpuForce-> %E\n", 
        //    gpuAlgo->getAlgoName(), refAlgo->getAlgoName(), max ); fflush( stdout );
        refAlgo->integrateGravitation( g_dt, g_damping );
    }

    gpuAlgo->integrateGravitation( g_dt, g_damping );

    if ( g_bGPUCrossCheck && g_fGPUCrosscheckInput ) {
        if ( memcmp( g_hostAOS_Force.data(), g_hostAOS_Force_Golden.data(), 3*g_N*sizeof(float) ) ) {
            printf( "GPU CROSSCHECK FAILURE: Disagreement with golden values\n" );
            goto Error;
        }
    }

    return true;
Error:
    return false;
}

workerThread *g_CPUThreadPool;
int g_numCPUCores;

workerThread *g_GPUThreadPool;
int g_numGPUs;

struct gpuInit_struct
{
    int iGPU;

    hipError_t status;
};

void
initializeGPU( void *_p )
{
    hipError_t status;

    gpuInit_struct *p = (gpuInit_struct *) _p;
    cuda(SetDevice( p->iGPU ) );
    cuda(SetDeviceFlags( hipDeviceMapHost ) );
    cuda(Free(0) );
Error:
    p->status = status;    
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    // kiloparticles
    int kParticles = 4, kMaxIterations = 0;
    NBodyAlgorithm<float> *refAlgo = nullptr;
    NBodyAlgorithm<float> *gpuAlgo = nullptr;
    int seed = 7;

    if ( 1 == argc ) {
        printf( "Usage: nbody --numbodies <N> [--nocpu] [--nocrosscheck] [--iterations <N>]\n" );
        printf( "    --numbodies is multiplied by 1024 (default is 4)\n" );
        printf( "    By default, the app checks results against a CPU implementation; \n" );
        printf( "    disable this behavior with --nocrosscheck.\n" );
        printf( "    The CPU implementation may be disabled with --nocpu.\n" );
        printf( "    --nocpu implies --nocrosscheck.\n\n" );
        printf( "    --nosimd uses serial CPU implementation instead of SIMD.\n" );
        printf( "    --iterations specifies a fixed number of iterations to execute\n");
        return 1;
    }

        g_numCPUCores = processorCount();
#if 0
    {
        g_CPUThreadPool = new workerThread[g_numCPUCores];
        for ( size_t i = 0; i < g_numCPUCores; i++ ) {
            if ( ! g_CPUThreadPool[i].initialize( ) ) {
                fprintf( stderr, "Error initializing thread pool\n" );
                return 1;
            }
        }
    }
#endif

    status = hipGetDeviceCount( &g_numGPUs );
    g_bCUDAPresent = (hipSuccess == status) && (g_numGPUs > 0);
    if ( g_bCUDAPresent ) {
        hipDeviceProp_t prop;
        cuda(GetDeviceProperties( &prop, 0 ) );
        g_bSM30Present = prop.major >= 3;
    }
    else {
        fprintf( stderr, "nbody: no GPUs\n" );
        exit(1);
    }
    g_bNoCPU = chCommandLineGetBool( "nocpu", argc, argv );
    if ( g_bNoCPU && ! g_bCUDAPresent ) {
        printf( "--nocpu specified, but no CUDA present...exiting\n" );
        exit(1);
    }

    g_bCrossCheck = ! chCommandLineGetBool( "nocrosscheck", argc, argv );
    if ( g_bNoCPU ) {
        g_bCrossCheck = false;
    }
    if ( g_bCrossCheck && chCommandLineGetBool( "nosse", argc, argv ) ) {
        g_bUseSIMDForCrossCheck = false;
    }

    chCommandLineGet( &kParticles, "numbodies", argc, argv );
    g_N = kParticles*1024;

    chCommandLineGet( &kMaxIterations, "iterations", argc, argv);

    // Round down to the nearest multiple of the CPU count (e.g. if we have
    // a system with a CPU count that isn't a power of two, we need to round)
    g_N -= g_N % g_numCPUCores;

    if ( chCommandLineGetBool( "gpu-crosscheck", argc, argv ) ) {
        g_bGPUCrossCheck = true;
    }
    g_bGPUCrossCheck = chCommandLineGetBool( "gpu-crosscheck", argc, argv );
#if 0
    {
        char *szFilename;
        if ( chCommandLineGet( &szFilename, "gpu-crosscheck-input-file", argc, argv ) ) {
            if ( ! g_bGPUCrossCheck ) {
                fprintf( stderr, "GPU crosscheck input file requires --gpu-crosscheck\n" );
                goto Error;
            }
            g_fGPUCrosscheckInput = fopen( szFilename, "rb" );
            if ( ! g_fGPUCrosscheckInput ) {
                fprintf( stderr, "Could not open %s for input\n", szFilename );
                goto Error;
            }
            {
                int version;
                if ( 1 != fread( &version, sizeof(int), 1, g_fGPUCrosscheckInput ) ) {
                    fprintf( stderr, "Read of version failed\n" );
                    goto Error;
                }
                if ( version != NBODY_GOLDENFILE_VERSION ) {
                    fprintf( stderr, "File version mismatch - generate new golden files!\n" );
                    goto Error;
                }
            }
            if ( 1 != fread( &g_N, sizeof(int), 1, g_fGPUCrosscheckInput ) ) {
                fprintf( stderr, "Read of particle count failed\n" );
                goto Error;
            }
            if ( 1 != fread( &kMaxIterations, sizeof(int), 1, g_fGPUCrosscheckInput ) ) {
                fprintf( stderr, "Read of iteration count failed\n" );
                goto Error;
            }
            printf( "%d iterations specified in input file\n", kMaxIterations );
        }
        if ( chCommandLineGet( &szFilename, "gpu-crosscheck-output-file", argc, argv  ) ) {
            if ( g_fGPUCrosscheckInput ) {
                fprintf( stderr, "Crosscheck input and output files are mutually exclusive. Please specify only one.\n" );
                goto Error;
            }            
            if ( ! g_bGPUCrossCheck ) {
                fprintf( stderr, "GPU crosscheck output file requires --gpu-crosscheck\n" );
                goto Error;
            }
            g_fGPUCrosscheckOutput = fopen( szFilename, "wb" );
            if ( ! g_fGPUCrosscheckOutput ) {
                fprintf( stderr, "Could not open %s for output\n", szFilename );
                goto Error;
            }
            if ( ! kMaxIterations ) {
                fprintf( stderr, "Must specify --iterations when generating output file for GPU cross check.\n" );
                goto Error;
            }
            {
                int version = NBODY_GOLDENFILE_VERSION;
                if ( 1 != fwrite( &version, sizeof(int), 1, g_fGPUCrosscheckOutput ) ) {
                    fprintf( stderr, "Write of version failed\n" );
                    goto Error;
                }
            }

            if ( 1 != fwrite( &g_N, sizeof(int), 1, g_fGPUCrosscheckOutput ) ) {
                fprintf( stderr, "Write of particle count failed\n" );
                goto Error;
            }
            if ( 1 != fwrite( &kMaxIterations, sizeof(int), 1, g_fGPUCrosscheckOutput ) ) {
                fprintf( stderr, "Write of iteration count failed\n" );
                goto Error;
            }
        }
    }

    chCommandLineGet( &g_ZeroThreshold, "zero", argc, argv );

    if ( g_numGPUs ) {
        // optionally override GPU count from command line
        chCommandLineGet( &g_numGPUs, "numgpus", argc, argv );
        g_GPUThreadPool = new workerThread[g_numGPUs];
        for ( size_t i = 0; i < g_numGPUs; i++ ) {
            if ( ! g_GPUThreadPool[i].initialize( ) ) {
                fprintf( stderr, "Error initializing thread pool\n" );
                return 1;
            }
        }
        for ( int i = 0; i < g_numGPUs; i++ ) {
            gpuInit_struct initGPU = {i};
            g_GPUThreadPool[i].delegateSynchronous( 
                initializeGPU, 
                &initGPU );
            if ( hipSuccess != initGPU.status ) {
                fprintf( stderr, "Initializing GPU %d failed "
                    " with %d (%s)\n",
                    i, 
                    initGPU.status, 
                    hipGetErrorString( initGPU.status ) );
                return 1;
            }
        }
    }

    printf( "Running simulation with %d particles, crosscheck %s, CPU %s\n", (int) g_N,
        g_bCrossCheck ? "enabled" : "disabled",
        g_bNoCPU ? "disabled" : "enabled" );

    g_maxAlgorithm = CPU_AOS;
#endif

#if 0
#if defined(HAVE_SIMD_OPENMP)
    g_maxAlgorithm = CPU_SIMD_openmp;
#elif defined(HAVE_SIMD_THREADED)
    g_maxAlgorithm = CPU_SIMD_threaded;
#elif defined(HAVE_SIMD)
    g_maxAlgorithm = CPU_SIMD;
#else
    g_maxAlgorithm = CPU_SOA;
#endif
    g_Algorithm = g_bCUDAPresent ? GPU_AOS : g_maxAlgorithm;
	g_Algorithm = multiGPU_SingleCPUThread;
    if ( g_bCUDAPresent || g_bNoCPU ) {
        // max algorithm is different depending on whether SM 3.0 is present
        g_maxAlgorithm = g_bSM30Present ? GPU_AOS_tiled_const : multiGPU_MultiCPUThread;
    }
#endif

    refAlgo = new NBodyAlgorithm<float>;
    if ( ! refAlgo->Initialize( g_N, seed, g_softening ) )
        goto Error;

    if ( g_bCUDAPresent ) {
        hipDeviceProp_t propForVersion;

        cuda(SetDeviceFlags( hipDeviceMapHost ) );
        cuda(GetDeviceProperties( &propForVersion, 0 ) );
        if ( propForVersion.major < 3 ) {
            // Only SM 3.x supports shuffle and fast atomics, so we cannot run
            // some algorithms on this board.
            g_maxAlgorithm = multiGPU_MultiCPUThread;
        }

        gpuAlgo = new NBodyAlgorithm_SSE<float>;
        //gpuAlgo = new NBodyAlgorithm_SOA<float>;
        //gpuAlgo = new NBodyAlgorithm_GPU<float>;
        if ( ! gpuAlgo->Initialize( g_N, seed, g_softening ) )
            goto Error;

#if 0
        cuda(HostAlloc( (void **) &g_hostAOS_PosMass, 4*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        for ( int i = 0; i < 3; i++ ) {
            cuda(HostAlloc( (void **) &g_hostSOA_Pos[i], g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
            cuda(HostAlloc( (void **) &g_hostSOA_Force[i], g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        }
        cuda(HostAlloc( (void **) &g_hostAOS_Force, 3*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        cuda(HostAlloc( (void **) &g_hostAOS_Force_Golden, 3*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        cuda(HostAlloc( (void **) &g_hostAOS_VelInvMass, 4*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        cuda(HostAlloc( (void **) &g_hostSOA_Mass, g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        cuda(HostAlloc( (void **) &g_hostSOA_InvMass, g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );

        cuda(Malloc( &g_dptrAOS_PosMass, 4*g_N*sizeof(float) ) );
        cuda(Malloc( (void **) &g_dptrAOS_Force, 3*g_N*sizeof(float) ) );
#endif

        if ( g_bGPUCrossCheck  ) {
            printf( "GPU cross check enabled (%d GPUs), disabling CPU\n", g_numGPUs );
            g_bNoCPU = true;
            g_bCrossCheck = false;
            if ( g_numGPUs < 2 ) {
                fprintf( stderr, "GPU cross check enabled, but <2 GPUs available\n" );
                goto Error;
            }
            for ( int i = 0; i < g_numGPUs; i++ ) {
                cuda(HostAlloc( (void **) (&g_hostAOS_gpuCrossCheckForce[i]), 3*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
            }
        }
    }
    else {
        g_hostAOS_PosMass = std::vector<PosMass<float>>( g_N );
        for ( int i = 0; i < 3; i++ ) {
            g_hostSOA_Pos[i] = new float[g_N];
            g_hostSOA_Force[i] = new float[g_N];
        }
        g_hostSOA_Mass = new float[g_N];
        g_hostAOS_Force = std::vector<Force3D<float>>( g_N );//new float[3*g_N];
        g_hostAOS_Force_Golden = std::vector<Force3D<float>>( g_N );//new float[3*g_N];
        g_hostAOS_VelInvMass = std::vector<VelInvMass<float>>( g_N ) ;//new float[4*g_N];
        g_hostSOA_Mass = new float[g_N];
        g_hostSOA_InvMass = new float[g_N];
    }

#if 0
    randomUnitBodies( seed, g_hostAOS_PosMass, g_hostAOS_VelInvMass, g_N );
    for ( size_t i = 0; i < g_N; i++ ) {
        g_hostSOA_Mass[i] = g_hostAOS_PosMass[i].mass_;
        g_hostSOA_InvMass[i] = 1.0f / g_hostSOA_Mass[i];
    }

    // gather performance data over GPU implementations
    // for different problem sizes.

    printf( "kBodies\t" );
    for ( int algorithm = GPU_AOS; 
              algorithm < sizeof(rgszAlgorithmNames)/sizeof(rgszAlgorithmNames[0]); 
              algorithm++ ) {
        printf( "%s\t", rgszAlgorithmNames[algorithm] );
    }
    printf( "\n" );

    for ( int kBodies = 3; kBodies <= 96; kBodies += 3 ) {

	g_N = 1024*kBodies;

        printf( "%d\t", kBodies );

	for ( int algorithm = GPU_AOS; 
                  algorithm < sizeof(rgszAlgorithmNames)/sizeof(rgszAlgorithmNames[0]); 
                  algorithm++ ) {
            float sum = 0.0f;
            const int numIterations = 10;
            for ( int i = 0; i < numIterations; i++ ) {
                float ms, err;
		if ( ! ComputeGravitation( &ms, &err, (nbodyAlgorithm_enum) algorithm, g_bCrossCheck ) ) {
			fprintf( stderr, "Error computing timestep\n" );
			exit(1);
		}
                sum += ms;
            }
            sum /= (float) numIterations;

            double interactionsPerSecond = (double) g_N*g_N*1000.0f / sum;
            if ( interactionsPerSecond > 1e9 ) {
                printf ( "%.2f\t", interactionsPerSecond/1e9 );
            }
            else {
                printf ( "%.3f\t", interactionsPerSecond/1e9 );               
            }
        }
        printf( "\n" );
    }
    return 0;
#endif
    {
        int kIterations = 0;
        bool bStop = false;
        while ( ! bStop ) {
            float ms, err;

            if ( ! ComputeGravitation( &ms, &err, refAlgo, gpuAlgo, g_bCrossCheck ) ) {
                fprintf( stderr, "Error computing timestep\n" );
                exit(1);
            }
            double interactionsPerSecond = (double) g_N*g_N*1000.0f / ms;
            if ( interactionsPerSecond > 1e9 ) {
                printf ( "\r%s: %8.2f ms = %8.3fx10^9 interactions/s (Abs. error: %E)\n",
                    gpuAlgo->getAlgoName(),
                    ms, 
                    interactionsPerSecond/1e9, 
                    err );
            }
            else {
                printf ( "\r%s: %8.2f ms = %8.3fx10^6 interactions/s (Abs. error: %E)\n",
                    gpuAlgo->getAlgoName(),
                    ms, 
                    interactionsPerSecond/1e6, 
                    err );
            }
            if (kMaxIterations) {
                kIterations++;
                if (kIterations >= kMaxIterations) {
                    bStop = true;
                }
            }
            if ( kbhit() ) {
                char c = getch();
                switch ( c ) {
                    case ' ':
                        if ( g_Algorithm == g_maxAlgorithm ) {
                            g_Algorithm = g_bNoCPU ? GPU_AOS : CPU_AOS;
                            // Skip slow CPU implementations if we are using SIMD for cross-check
                            if ( g_bUseSIMDForCrossCheck ) {
#if defined(HAVE_SIMD_THREADED)
                                g_Algorithm = CPU_SIMD_threaded;
#elif defined(HAVE_SIMD_OPENMP)
                                g_Algorithm = CPU_SIMD_openmp;
#endif
                            }
                        }
                        else {
                            g_Algorithm = (enum nbodyAlgorithm_enum) (g_Algorithm+1);
                        }
                        break;
                    case 'q':
                    case 'Q':
                        bStop = true;
                        break;
                }

            }
        }
    }

    if ( g_fGPUCrosscheckInput ) fclose( g_fGPUCrosscheckInput );
    if ( g_fGPUCrosscheckOutput ) fclose( g_fGPUCrosscheckOutput );

    return 0;
Error:
    if ( g_fGPUCrosscheckInput ) fclose( g_fGPUCrosscheckInput );
    if ( g_fGPUCrosscheckOutput ) fclose( g_fGPUCrosscheckOutput );
    if ( hipSuccess != status ) {
        printf( "CUDA Error: %s\n", hipGetErrorString( status ) );
    }
    return 1;
}
