#include "hip/hip_runtime.h"
/*
 *
 * eventRecord.cu
 *
 * Microbenchmark for throughput of event recording
 *
 * Build with: nvcc -I ../chLib <options> eventRecord.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2014, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chError.h"
#include "chTimer.h"

__global__
void
NullKernel()
{
}

#define EVENTRECORD_LAUNCH   0x01
#define EVENTRECORD_BLOCKING 0x02

template<int Flags>
double
usPerLaunch( int cIterations, int cEvents )
{
    hipError_t status;
    double microseconds, ret;
    hipEvent_t *events = new hipEvent_t[cEvents];
    chTimerTimestamp start, stop;

    if ( ! events ) goto Error;
    memset( events, 0, cEvents*sizeof(hipEvent_t) );
    for ( int i = 0; i < cEvents; i++ ) {
        cuda(EventCreateWithFlags(  &events[i], (Flags & EVENTRECORD_BLOCKING) ? hipEventBlockingSync : 0 ) );
    }

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
        if ( Flags & EVENTRECORD_LAUNCH) NullKernel<<<1,1>>>();
        for ( int j = 0; j < cEvents; j++ ) {
            cuda(EventRecord( events[j], NULL ) );
        }
    }
    cuda(DeviceSynchronize() );
    chTimerGetTime( &stop );

    microseconds = 1e6*chTimerElapsedTime( &start, &stop );
    if ( cEvents ) cIterations *= cEvents;
    ret = microseconds / (float) cIterations;

Error:
    if ( events ) {
        for ( int i = 0; i < cEvents; i++ ) {
            hipEventDestroy( events[i] );
        }
    }
    delete[] events;
    return (status) ? 0.0 : ret;
}

int
main( int argc, char *argv[] )
{
    hipFree( 0 );
    const int cIterations = 10000;
    printf( "Measuring blocking event record overhead...\n" ); fflush( stdout );

    printf( "#events\tus per event signaling\n" );
    for ( int cEvents = 0; cEvents < 5; cEvents += 1 ) {
        printf( "%d\t%.2f\n", cEvents*10, usPerLaunch<EVENTRECORD_BLOCKING>(cIterations, cEvents) );
    }
    printf( "Measuring asynchronous launch+event signaling...\n" ); fflush( stdout );
    for ( int cEvents = 0; cEvents < 5; cEvents += 1 ) {
        printf( "%d\t%.2f\n", cEvents*10, usPerLaunch<EVENTRECORD_LAUNCH | EVENTRECORD_BLOCKING>(cIterations, cEvents) );
    }

    return 0;
}

