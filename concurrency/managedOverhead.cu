#include "hip/hip_runtime.h"
/*
 *
 * managedOverhed.cu
 *
 * Microbenchmark to measure overhead of managed memory.
 * This app illustrates how managed memory coherency appears to be
 * implemented with paging.  Only pages accessed by the host
 * code get copied from device to host.
 *
 * Interestingly, the CUDA driver also does not appear to be
 * doing dirty-bit optimizations, since the null kernel is not
 * actually touching any GPU memory.
 *
 * Build with: nvcc -I ../chLib <options> managedOverhead.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2014, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce thce above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chError.h"
#include "chTimer.h"

__global__
void
NullKernel()
{
}

const size_t pageSize = 4096;

template<bool bTouch>
double
usPerLaunch( int cIterations, size_t cPages=0 )
{
    hipError_t status;
    double microseconds, ret;
    chTimerTimestamp start, stop;
    void *p = 0;

    cuda(Free(0) );
    if ( cPages ) {
        cuda(MallocManaged( &p, cPages*pageSize ) );
    }

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
        NullKernel<<<1,1>>>();
        cuda(DeviceSynchronize() );
        if ( bTouch && 0 != p ) {
            for ( int iPage = 0; iPage < cPages; iPage++ ) {
                ((volatile unsigned char *) p)[iPage*pageSize] |= 1;
            }
        }
    }
    chTimerGetTime( &stop );

    microseconds = 1e6*chTimerElapsedTime( &start, &stop );
    ret = microseconds / (float) cIterations;
    hipFree( p );
Error:
    return (status) ? 0.0 : ret;
}

int
main( int argc, char *argv[] )
{
    const int cIterations = 1000;
    printf( "Measuring synchronous launch time...\n" ); fflush( stdout );

    printf( "%8.2f us (0 pages)\n", usPerLaunch<false>(cIterations) );
    for ( size_t cPages = 1; cPages < 8192; cPages *= 2 ) {
        printf( "%8.2f us (%d pages)\n", usPerLaunch<true>(cIterations, cPages), cPages );
    }
    printf( "Without touching memory:\n" );
    for ( size_t cPages = 1; cPages < 8192; cPages *= 2 ) {
        printf( "%8.2f us (%d pages)\n", usPerLaunch<false>(cIterations, cPages), cPages );
    }

    return 0;
}
