#include "hip/hip_runtime.h"
/*
 *
 * nullKernelAsyncGraph.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsyncGraph.cu
 * Requires: CUDA graph availability.
 *
 * Copyright (c) 2023, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "chError.h"
#include "chTimer.h"

constexpr int itersPerGraph = 100;

__global__
void
NullKernel()
{
}

hipError_t
cudaCreateGraphNullKernelLaunches( hipGraph_t *graph, hipGraphExec_t *graphInstance, hipStream_t stream, int cIterations )
{
    hipError_t status;

    cuda(StreamBeginCapture(stream, hipStreamCaptureModeGlobal));
    for ( int i = 0; i < cIterations; ++i ) {
        NullKernel<<<1,1,0,stream>>>();
    }
    cuda(StreamEndCapture(stream, graph));
    cuda(GraphInstantiate(graphInstance, *graph, NULL, NULL, 0));
    return hipSuccess;
Error:
    return status;
}

double
usPerLaunch( int cIterations )
{
    hipError_t status;
    double microseconds, ret;
    hipStream_t stream;
    hipGraph_t graph;
    hipGraphExec_t graphInstance;
    chTimerTimestamp start, stop;

    cuda(Free(0));
    cuda(StreamCreate( &stream ));
    cuda(CreateGraphNullKernelLaunches( &graph, &graphInstance, stream, itersPerGraph ));

    chTimerGetTime( &start );
    int i;
    for ( i = 0; i < cIterations; i += itersPerGraph ) {
        cuda(GraphLaunch( graphInstance, NULL ));
    }
    cuda(DeviceSynchronize());
    chTimerGetTime( &stop );

    microseconds = 1e6*chTimerElapsedTime( &start, &stop );
    ret = microseconds / (float) i;

Error:
    return (status) ? 0.0 : ret;
}

int
main( int argc, char *argv[] )
{
    const int cIterations = 100000;
    printf( "Measuring asynchronous launch time (launched w graphs)... " ); fflush( stdout );

    printf( "%.2f us\n", usPerLaunch(cIterations) );

    return 0;
}
