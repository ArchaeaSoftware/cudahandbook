#include "hip/hip_runtime.h"
/*
 *
 * breakevenKernelAsync.cu
 *
 * Microbenchmark of kernel launch overhead for kernels that
 * do varying amounts of work.
 *
 * Build with: nvcc -I ../chLib <options> breakevenKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chTimer.h"

__device__ int deviceTime;

__global__
void
WaitKernel( int cycles, bool bWrite )
{
    int start = clock();
    int stop;
    do {
        stop = clock();
    } while ( stop - start < cycles );
    if ( bWrite && threadIdx.x==0 && blockIdx.x==0 ) {
        deviceTime = stop - start;
    }
}

int
main( int argc, char *argv[] )
{
    const int cIterations = 100000;

	// Take a warm-up lap
    chTimerTimestamp start, stop;
    for ( int i = 0; i < cIterations; i++ ) {
        WaitKernel<<<1,1>>>( 0, false );
    }
    hipDeviceSynchronize();

    printf("Cycles\tus\n" );
    for ( int cycles = 0; cycles < 2500; cycles += 100 ) {
        printf( "%d\t", cycles ); fflush( stdout );
        chTimerGetTime( &start );
        for ( int i = 0; i < cIterations; i++ ) {
            WaitKernel<<<1,1>>>( cycles, false );
        }
        hipDeviceSynchronize();
        chTimerGetTime( &stop );
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;

        printf( "%.2f\n", usPerLaunch );
    }


    return 0;
}
