#include "hip/hip_runtime.h"
/*
 *
 * nullKernelSync.cu
 *
 * Microbenchmark for throughput of synchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelSync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2014, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chError.h"
#include "chTimer.h"

__global__
void
NullKernel()
{
}

double
usPerLaunch( int cIterations )
{
    hipError_t status;
    double microseconds, ret;
    chTimerTimestamp start, stop;

    cuda(Free(0) );

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
        NullKernel<<<1,1>>>();
        cuda(DeviceSynchronize() );
    }
    chTimerGetTime( &stop );

    microseconds = 1e6*chTimerElapsedTime( &start, &stop );
    ret = microseconds / (float) cIterations;

Error:
    return (status) ? 0.0 : ret;
}

int
main( int argc, char *argv[] )
{
    const int cIterations = 100000;
    printf( "Measuring synchronous launch time... " ); fflush( stdout );

    printf( "%.2f us\n", usPerLaunch(cIterations) );

    return 0;
}
