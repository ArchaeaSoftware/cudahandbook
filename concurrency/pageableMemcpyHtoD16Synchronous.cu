#include "hip/hip_runtime.h"
/*
 *
 * pageableMemcpyHtoDSynchronous.cu
 *
 * Microdemo that illustrates how necessary CPU/GPU concurrency
 * is for a good-performance pageable memcpy.  Identical to
 * pageableMemcpyHtoD.cu except the event synchronize is in a
 * place that breaks concurrency between the CPU and GPU.
 *
 * A pair of pinned staging buffers are allocated, and after the first
 * staging buffer has been filled, the GPU pulls from one while the
 * CPU fills the other.  CUDA events are used for synchronization.
 *
 * This implementation uses the SSE-optimized memcpy of memcpy16.cpp,
 * so for simplicity, it requires host pointers to be 16-byte aligned.
 *
 * Build with: nvcc -I ../chLib <options> pageableMemcpyHtoD16Synchronous.cu memcpy16.cpp
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chError.h"
#include "chTimer.h"

#define STAGING_BUFFER_SIZE 1048576

void *g_hostBuffers[2];
hipEvent_t g_events[2];

// these are already defined on some platforms - make our
// own definitions that will work.
#undef min
#undef max
#define min(a,b) ((a)<(b)?(a):(b))
#define max(a,b) ((b)<(a)?(a):(b))

extern bool memcpy16( void *_dst, const void *_src, size_t N );

void
chMemcpyHtoD( void *device, const void *host, size_t N ) 
{
    hipError_t status;
    char *dst = (char *) device;
    const char *src = (const char *) host;
    int stagingIndex = 0;
    while ( N ) {
        size_t thisCopySize = min( N, STAGING_BUFFER_SIZE );

//        cuda(EventSynchronize( g_events[stagingIndex] ) );
        memcpy16( g_hostBuffers[stagingIndex], src, thisCopySize ); 
        cuda(MemcpyAsync( dst, g_hostBuffers[stagingIndex], thisCopySize, 
            hipMemcpyHostToDevice, NULL ) );
        cuda(EventRecord( g_events[1-stagingIndex], NULL ) );
        cuda(EventSynchronize( g_events[1-stagingIndex] ) );
        dst += thisCopySize;
        src += thisCopySize;
        N -= thisCopySize;
        stagingIndex = 1 - stagingIndex;
    }
Error:
    return;
}

bool
TestMemcpy( int *dstDevice, int *srcHost, const int *srcOriginal,
            size_t dstOffset, size_t srcOffset, size_t numInts )
{
    chMemcpyHtoD( dstDevice+dstOffset, srcOriginal+srcOffset, numInts*sizeof(int) );
    hipMemcpy( srcHost, dstDevice+dstOffset, numInts*sizeof(int), hipMemcpyDeviceToHost );
    for ( size_t i = 0; i < numInts; i++ ) {
        if ( srcHost[i] != srcOriginal[srcOffset+i] ) {
            return false;
        }
    }
    return true;
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    int *deviceInt = 0;
    int *hostInt = 0;
    const size_t numInts = 32*1048576;
    const int cIterations = 10;
    int *testVector = 0;
    printf( "Pageable memcpy (16-byte aligned)... " ); fflush( stdout );

    chTimerTimestamp start, stop;

    cuda(HostAlloc( &g_hostBuffers[0], STAGING_BUFFER_SIZE, hipHostMallocDefault ) );
    cuda(HostAlloc( &g_hostBuffers[1], STAGING_BUFFER_SIZE, hipHostMallocDefault ) );
    cuda(EventCreate( &g_events[0] ) );
    cuda(EventRecord( g_events[0], 0 ) );  // so it is signaled on first synchronize
    cuda(EventCreate( &g_events[1] ) );
    cuda(EventRecord( g_events[1], 0 ) );  // so it is signaled on first synchronize

    cuda(Malloc( &deviceInt, numInts*sizeof(int) ) );
    cuda(HostAlloc( &hostInt, numInts*sizeof(int), 0 ) );

    testVector = (int *) malloc( numInts*sizeof(int) );
    if ( ! testVector ) {
        printf( "malloc() failed\n" );
        return 1;
    }
    for ( size_t i = 0; i < numInts; i++ ) {
        testVector[i] = rand();
    }

    if ( ! TestMemcpy( deviceInt, hostInt, testVector, 0, 0, numInts ) ) {
        goto Error;
    }
    for ( int i = 0; i < cIterations; i++ ) {
        size_t numInts4 = numInts / 4;
        size_t dstOffset = rand() % (numInts4-1);
        size_t srcOffset = rand() % (numInts4-1);
        size_t intsThisIteration = 1 + rand() % (numInts4-max(dstOffset,srcOffset)-1);
        dstOffset *= 4;
        srcOffset *= 4;
        intsThisIteration *= 4;
        if ( ! TestMemcpy( deviceInt, hostInt, testVector, dstOffset, srcOffset, intsThisIteration ) ) {
            TestMemcpy( deviceInt, hostInt, testVector, dstOffset, srcOffset, intsThisIteration );
            goto Error;
        }
    }

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
        chMemcpyHtoD( deviceInt, testVector, numInts*sizeof(int) ) ;
    }
    cuda(DeviceSynchronize() );
    chTimerGetTime( &stop );

    {
        double MBytes = cIterations*numInts*sizeof(int) / 1048576.0;
        double MBpers = MBytes / chTimerElapsedTime( &start, &stop );

        printf( "%.2f MB/s\n", MBpers );
    }

    hipFree( deviceInt );
    hipHostFree( hostInt );
    return 0;
Error:
    printf( "Error\n" );
    return 1;
}
