#include "hip/hip_runtime.h"
/*
 *
 * divergence.cu
 *
 * Microdemo to measure performance implications of conditional code.
 *
 * Build with: nvcc [--gpu-architecture sm_xx] [-D USE_FLOAT] [-D USE_IF_STATEMENT] divergence.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2021, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <unistd.h>
#include <chError.h>

//
// parameters p and n are expected to account for grid structure
// apply threadIdx and/or blockIdx to p before passing in;
// apply blockDim and/or gridDim to n before passing in.
//
template<int base>
void __device__ sumFloats( float *p, size_t N, size_t n )
{
    float f = base;
    for ( size_t i = 0; i < N; i++ ) {
        *p += f;
        p += n;
    }
}

typedef void(*psumFloats)(float *, size_t, size_t);

__device__ psumFloats rgSumFloats[] = {
    sumFloats< 0>, sumFloats< 1>, sumFloats< 2>, sumFloats< 3>,
    sumFloats< 4>, sumFloats< 5>, sumFloats< 6>, sumFloats< 7>,
    sumFloats< 8>, sumFloats< 9>, sumFloats<10>, sumFloats<11>,
    sumFloats<12>, sumFloats<13>, sumFloats<14>, sumFloats<15>,
    sumFloats<16>, sumFloats<17>, sumFloats<18>, sumFloats<19>,
    sumFloats<20>, sumFloats<21>, sumFloats<22>, sumFloats<23>,
    sumFloats<24>, sumFloats<25>, sumFloats<26>, sumFloats<27>,
    sumFloats<28>, sumFloats<29>, sumFloats<30>, sumFloats<31> };

__global__ void
sumFloats_bywarp( float *p, size_t N )
{
    uint32_t warpid = threadIdx.x>>5;
    N /= blockDim.x*gridDim.x;
    rgSumFloats[warpid]( p+threadIdx.x+blockIdx.x*blockDim.x, N, blockDim.x*gridDim.x );
}

__global__ void
sumFloats_bythread( float *p, size_t N )
{
    
}

int
main()
{
    hipError_t status;
    size_t N = 1024*1024*1024UL;
    float *p = 0;
    float et;
    hipEvent_t start = 0, stop = 0;

    cuda(Malloc( (void **) &p, N*sizeof(float)) );
    cuda(Memset( p, 0, N*sizeof(float)) );
    cuda(EventCreate( &start ));
    cuda(EventCreate( &stop ));

    cuda(EventRecord( start ));
    sumFloats_bywarp<<<3072,256>>>( p, N );
    cuda(EventRecord( stop ));
    cuda(DeviceSynchronize());
    cuda(EventElapsedTime( &et, start, stop ));
    
    printf( "%.2f ms = %.2f Gops/s\n", et, (double) N*1000.0/et/1e9 );

    hipFree( p );
    hipEventDestroy( stop );
    hipEventDestroy( start );
    return 0;
Error:
    return 1;
}
