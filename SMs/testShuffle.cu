#include "hip/hip_runtime.h"
/*
 *
 * testShuffle.cu
 *
 * Microdemo to illustrate the workings of Kepler's new shuffle instruction.
 * 
 * Build with: nvcc -I ..\chLib <options> testShuffle.cu
 * Requires: SM 3.0 or higher.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <chError.h>

#include <stdio.h>

#include <sm_30_intrinsics.h>

__global__ void
TestShuffle( int *out, const int *in, size_t N  )
{
    size_t i = blockIdx.x*blockDim.x+threadIdx.x;

    int value = (int) i;//in[i];
    out[i] = __shfl_up_sync( 0xffffffff, value, 1 );
}

hipError_t
PrintShuffle( int offset, size_t cInts )
{
    int *dptr = 0;
    hipError_t status;
    int h[64];
    cuda(Malloc( &dptr, cInts*sizeof(int) ) );
    TestShuffle<<<1,cInts>>>( dptr, dptr, cInts );
    cuda(Memcpy( h, dptr, cInts*sizeof(int), hipMemcpyDeviceToHost ) );
    for ( size_t i = 0; i < cInts; i++ ) {
        printf( "%3x", h[i] );
        if (31==i%32) printf("\n");
    }
    printf( "\n" );
Error:
    hipFree( dptr );
    return status;
}

int
main( int argc, char *argv[] )
{
    int ret = 1;
    int cInts = 64;
    hipError_t status;

    CUDART_CHECK( PrintShuffle( 1, cInts ) );
    return 0;
Error:
    printf( "Error %d (%s)\n", status, hipGetErrorString( status ) );
    return ret;
}
