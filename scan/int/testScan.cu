#include "hip/hip_runtime.h"
/*
 *
 * testScan.cu
 *
 * Microdemo to test scan algorithms.
 *
 * Build with: nvcc -I ..\chLib <options> testScan.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <chTimer.h>
#include <chAssert.h>
#include <chError.h>

#include "scanWarp.cuh"
#include "scanBlock.cuh"

#include "scanZeroPad.cuh"

#define min(a,b) ((a)<(b)?(a):(b))

enum ScanType {
    Inclusive, Exclusive
};

#include "scanFan.cuh"
#include "scanReduceThenScan.cuh"
#include "scanReduceThenScan_0.cuh"
#include "scan2Level.cuh"
#include "scanThrust.cuh"

void
ScanExclusiveCPU( int *out, const int *in, size_t N )
{
    int sum = 0;
    for ( size_t i = 0; i < N; i++ ) {
        int next = in[i]; // in case we are doing this in place
        out[i] = sum;
        sum += next;
    }
}

int
ScanInclusiveCPU( int *out, const int *in, size_t N )
{
    int sum = 0;
    for ( size_t i = 0; i < N; i++ ) {
        sum += in[i];
        out[i] = sum;
    }
    return sum;
}

void
RandomArray( int *out, size_t N, int modulus )
{
    for ( size_t i = 0; i < N; i++ ) {
        out[i] = rand() % modulus;
    }
}

template<class T>
bool
TestScan( const char *szScanFunction, 
          void (*pfnScanGPU)(T *, const T *, size_t, int), 
          size_t N, 
          int numThreads )
{
    bool ret = false;
    hipError_t status;
    int *inGPU = 0;
    int *outGPU = 0;
    int *inCPU = (T *) malloc( N*sizeof(T) );
    int *outCPU = (int *) malloc( N*sizeof(T) );
    int *hostGPU = (int *) malloc( N*sizeof(T) );
    if ( 0==inCPU || 0==outCPU || 0==hostGPU )
        goto Error;

    printf( "Testing %s (%d integers, %d threads/block)\n", 
        szScanFunction,
        (int) N,
        numThreads );

    cuda(Malloc( &inGPU, N*sizeof(T) ) );
    cuda(Malloc( &outGPU, N*sizeof(T) ) );
    cuda(Memset( inGPU, 0, N*sizeof(T) ) );
    cuda(Memset( outGPU, 0, N*sizeof(T) ) );

    cuda(Memset( outGPU, 0, N*sizeof(T) ) );

    RandomArray( inCPU, N, 256 );
for ( int i = 0; i < N; i++ ) {
    inCPU[i] = i;
}
    
    ScanInclusiveCPU( outCPU, inCPU, N );

    cuda(Memcpy( inGPU, inCPU, N*sizeof(T), hipMemcpyHostToDevice ) );
    pfnScanGPU( outGPU, inGPU, N, numThreads );
    cuda(Memcpy( hostGPU, outGPU, N*sizeof(T), hipMemcpyDeviceToHost ) );
    for ( size_t i = 0; i < N; i++ ) {
        if ( hostGPU[i] != outCPU[i] ) {
            printf( "Scan failed\n" );
#ifdef _WIN32
            __debugbreak();
#else
            assert(0);
#endif
            goto Error;
        }
    }
    ret = true;
Error:
    hipFree( outGPU );
    hipFree( inGPU );
    free( inCPU );
    free( outCPU );
    free( hostGPU );
    return ret;
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    int maxThreads;

    cuda(SetDevice( 0 ) );
    cuda(SetDeviceFlags( hipDeviceMapHost ) );

    {
        hipDeviceProp_t prop;
        cuda(GetDeviceProperties( &prop, 0 ));
        maxThreads = prop.maxThreadsPerBlock;
    }

#define SCAN_TEST_VECTOR( Function, N, numThreads ) do { \
    srand(0); \
    bool bSuccess = TestScan<int>( #Function, Function, N, numThreads ); \
    if ( ! bSuccess ) { \
        printf( "%s failed: N=%d, numThreads=%d\n", #Function, N, numThreads ); \
        exit(1); \
    } \
} while (0)

    for ( int numThreads = 256; numThreads <= maxThreads; numThreads *= 2 ) {
        
        for ( int numInts = 256; numInts <= 2048; numInts += 128 ) {

            SCAN_TEST_VECTOR( scan2Level<int>, numInts, numThreads );

            SCAN_TEST_VECTOR( scanFan<int>, numInts, numThreads );
            SCAN_TEST_VECTOR( scanReduceThenScan<int>, numInts, numThreads );
            SCAN_TEST_VECTOR( scanReduceThenScan_0<int>, numInts, numThreads );
            SCAN_TEST_VECTOR( scan2Level<int>, numInts, numThreads );
            SCAN_TEST_VECTOR( scan2Level_0<int>, numInts, numThreads );
        }

        for ( int numInts = 33*1048576-1; numInts < 33*1048576+1; numInts++ ) {

            SCAN_TEST_VECTOR( scan2Level<int>, numInts, numThreads );
            SCAN_TEST_VECTOR( scan2Level_0<int>, numInts, numThreads );

            SCAN_TEST_VECTOR( scanFan<int>, numInts, numThreads );
            SCAN_TEST_VECTOR( scanReduceThenScan<int>, numInts, numThreads );
            SCAN_TEST_VECTOR( scanReduceThenScan_0<int>, numInts, numThreads );

            SCAN_TEST_VECTOR( ScanThrust<int>, numInts, numThreads );
        }

    }
    return 0;
Error:
    return 1;
}
