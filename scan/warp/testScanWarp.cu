#include "hip/hip_runtime.h"
/*
 *
 * testScanWarp.cu
 *
 * Microdemo to test warp scan algorithms.
 *
 * Build with: nvcc -I ..\chLib <options> testScanWarp.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <chAssert.h>
#include <chCommandLine.h>
#include <chError.h>

#include "scanBlock.cuh" 
#include "scanWarp.cuh"
#include "scanWarp2.cuh"
#include "scanWarpShuffle.cuh"

#include "scanZeroPad.cuh"


#define min(a,b) ((a)<(b)?(a):(b))

enum ScanType {
    Inclusive, Exclusive
};

#include "scanFan.cuh"
#include "scanReduceThenScan.cuh"
#include "scanReduceThenScan_0.cuh"
#include "scan2Level.cuh"
#include "scanThrust.cuh"

template<int period>
void
ScanExclusiveCPUPeriodic( int *out, const int *in, size_t N )
{
    for ( size_t i = 0; i < N; i += period ) {
        int sum = 0;
        for ( size_t j = 0; j < period; j++ ) {
            int next = in[i+j]; // in case we are doing this in place
            out[i+j] = sum;
            sum += next;
        }
    }
}

template<int period>
void
ScanInclusiveCPUPeriodic( int *out, const int *in, size_t N )
{
    for ( size_t i = 0; i < N; i += period ) {
        int sum = 0;
        for ( size_t j = 0; j < period; j++ ) {
            sum += in[i+j];
            out[i+j] = sum;
        }
    }
}

template<ScanType scantype>
void
ScanCPU32( int *out, const int *in, size_t N )
{
    switch ( scantype ) {
        case Exclusive: return ScanExclusiveCPUPeriodic<32>( out, in, N );
        case Inclusive: return ScanInclusiveCPUPeriodic<32>( out, in, N );
    }
}

void
RandomArray( int *out, size_t N, int modulus )
{
    for ( size_t i = 0; i < N; i++ ) {
        out[i] = rand() % modulus;
    }
}

template<ScanType scantype>
__global__ void
ScanGPUWarp( int *out, const int *in, size_t N )
{
    extern __shared__ int sPartials[];
    for ( size_t i = blockIdx.x*blockDim.x;
                 i < N;
                 i += blockDim.x ) {
        sPartials[threadIdx.x] = in[i+threadIdx.x];
        __syncthreads();
        if ( scantype == Inclusive ) {
            out[i+threadIdx.x] = scanWarp<int,false>( sPartials+threadIdx.x );
        }
        else {
            out[i+threadIdx.x] = scanWarpExclusive<int,false>( sPartials+threadIdx.x );
        }
    }
}

template<ScanType scantype>
void
ScanGPU( 
    int *out, 
    const int *in, 
    size_t N, 
    int cThreads )
{
    int cBlocks = (int) (N/150);
    if ( cBlocks > 150 ) {
        cBlocks = 150;
    }
    ScanGPUWarp<scantype><<<cBlocks, cThreads, cThreads*sizeof(int)>>>( 
        out, in, N );
}

__global__ void
ScanInclusiveGPUWarp_0( int *out, const int *in, size_t N )
{
    extern __shared__ int sPartials[];
    const int sIndex = scanSharedIndex<true>( threadIdx.x );

    sPartials[sIndex-16] = 0;

    for ( size_t i = blockIdx.x*blockDim.x;
                 i < N;
                 i += blockDim.x ) {
        sPartials[sIndex] = in[i+threadIdx.x];
        out[i+threadIdx.x] = scanWarp<int,true>( sPartials+sIndex );
    }
}

void
ScanInclusiveGPU_0( 
    int *out, 
    const int *in, 
    size_t N, 
    int cThreads )
{
    int cBlocks = (int) (N/150);
    if ( cBlocks > 150 ) {
        cBlocks = 150;
    }
    ScanInclusiveGPUWarp_0<<<cBlocks, 
        cThreads, 
        scanSharedMemory<int,true>(cThreads)>>>( 
        out, in, N );
}

__global__ void
ScanExclusiveGPUWarp_0( int *out, const int *in, size_t N )
{
    extern __shared__ int sPartials[];
    const int sIndex = scanSharedIndex<true>( threadIdx.x );

    sPartials[sIndex-16] = 0;

    for ( size_t i = blockIdx.x*blockDim.x;
                 i < N;
                 i += blockDim.x ) {
        sPartials[sIndex] = in[i+threadIdx.x];
        out[i+threadIdx.x] = scanWarpExclusive<int,true>( sPartials+sIndex );
    }
}

void
ScanExclusiveGPU_0( 
    int *out, 
    const int *in, 
    size_t N, 
    int cThreads )
{
    int cBlocks = (int) (N/150);
    if ( cBlocks > 150 ) {
        cBlocks = 150;
    }
    ScanExclusiveGPUWarp_0<<<cBlocks, 
        cThreads, 
        scanSharedMemory<int,true>(cThreads)>>>( 
        out, in, N );
}

__global__ void
ScanInclusiveGPUWarp2( int *out, const int *in, size_t N )
{
    extern __shared__ int sPartials[];
    for ( size_t i = blockIdx.x*blockDim.x;
                 i < N;
                 i += blockDim.x ) {
        sPartials[threadIdx.x] = in[i+threadIdx.x];
        __syncthreads();
        out[i+threadIdx.x] = scanWarp2<int,false>( sPartials+threadIdx.x );
    }
}

void
ScanInclusiveGPU2( 
    int *out, 
    const int *in, 
    size_t N, 
    int cThreads )
{
    int cBlocks = (int) (N/150);
    if ( cBlocks > 150 ) {
        cBlocks = 150;
    }
    ScanInclusiveGPUWarp2<<<cBlocks, cThreads, cThreads*sizeof(int)>>>( 
            out, in, N );
}

__global__ void
ScanExclusiveGPUWarp2( int *out, const int *in, size_t N )
{
    extern __shared__ int sPartials[];
    for ( size_t i = blockIdx.x*blockDim.x;
                 i < N;
                 i += blockDim.x ) {
        sPartials[threadIdx.x] = in[i+threadIdx.x];
        __syncthreads();
        out[i+threadIdx.x] = scanWarpExclusive2<int,false>( sPartials+threadIdx.x );
    }
}

void
ScanExclusiveGPU2( 
    int *out, 
    const int *in, 
    size_t N, 
    int cThreads )
{
    int cBlocks = (int) (N/150);
    if ( cBlocks > 150 ) {
        cBlocks = 150;
    }
    ScanExclusiveGPUWarp2<<<cBlocks, cThreads, cThreads*sizeof(int)>>>( 
            out, in, N );
}

template<ScanType scantype>
__global__ void
ScanGPUWarpShuffle( int *out, const int *in, size_t N )
{
    for ( size_t i = blockIdx.x*blockDim.x;
                 i < N;
                 i += blockDim.x ) {
        if ( scantype == Inclusive ) {
            out[i+threadIdx.x] = scanWarpShuffle<5>( in[i+threadIdx.x] );
        }
        else {
            out[i+threadIdx.x] = exclusive_scan_warp_shfl<5>( in[i+threadIdx.x] );
        }
    }
}

template<ScanType scantype>
void
ScanGPUShuffle( 
    int *out, 
    const int *in, 
    size_t N, 
    int cThreads )
{
    int cBlocks = (int) (N/150);
    if ( cBlocks > 150 ) {
        cBlocks = 150;
    }
    ScanGPUWarpShuffle<scantype><<<cBlocks, cThreads>>>( out, in, N );
}

template<class T>
bool
TestScanWarp( 
    float *pMelementspersecond,
    const char *szScanFunction, 
    void (*pfnScanCPU)(T *, const T *, size_t),
    void (*pfnScanGPU)(T *, const T *, size_t, int), 
    size_t N, 
    int numThreads )
{
    bool ret = false;
    hipError_t status;
    int *inGPU = 0;
    int *outGPU = 0;
    int *inCPU = (T *) malloc( N*sizeof(T) );
    int *outCPU = (int *) malloc( N*sizeof(T) );
    int *hostGPU = (int *) malloc( N*sizeof(T) );
    hipEvent_t evStart = 0, evStop = 0;
    if ( 0==inCPU || 0==outCPU || 0==hostGPU )
        goto Error;

    printf( "Testing %s (%d threads/block)\n", szScanFunction, numThreads );

    cuda(EventCreate( &evStart ) );
    cuda(EventCreate( &evStop ) );
    cuda(Malloc( &inGPU, N*sizeof(T) ) );
    cuda(Malloc( &outGPU, N*sizeof(T) ) );
    cuda(Memset( inGPU, 0, N*sizeof(T) ) );
    cuda(Memset( outGPU, 0, N*sizeof(T) ) );

    cuda(Memset( outGPU, 0, N*sizeof(T) ) );

    RandomArray( inCPU, N, 256 );
for ( int i = 0; i < N; i++ ) {
    inCPU[i] = i;
}
    
    pfnScanCPU( outCPU, inCPU, N );

    cuda(Memcpy( inGPU, inCPU, N*sizeof(T), hipMemcpyHostToDevice ) );
    cuda(EventRecord( evStart, 0 ) );
    pfnScanGPU( outGPU, inGPU, N, numThreads );
    cuda(EventRecord( evStop, 0 ) );
    cuda(Memcpy( hostGPU, outGPU, N*sizeof(T), hipMemcpyDeviceToHost ) );
    for ( size_t i = 0; i < N; i++ ) {
        if ( hostGPU[i] != outCPU[i] ) {
            printf( "Scan failed\n" );
#ifdef _WIN32
            __debugbreak();//_asm int 3
#else
            assert(0);
#endif
            goto Error;
        }
    }
    {
        float ms;
        cuda(EventElapsedTime( &ms, evStart, evStop ) );
        double Melements = N/1e6;
        *pMelementspersecond = 1000.0f*Melements/ms;
    }
    ret = true;
Error:
    hipEventDestroy( evStart );
    hipEventDestroy( evStop );
    hipFree( outGPU );
    hipFree( inGPU );
    free( inCPU );
    free( outCPU );
    free( hostGPU );
    return ret;
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    int maxThreads;
    int numInts = 16*1048576;

    cuda(SetDevice( 0 ) );
    cuda(SetDeviceFlags( hipDeviceMapHost ) );

    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties( &prop, 0 );
        maxThreads = prop.maxThreadsPerBlock;
    }

#define SCAN_TEST_VECTOR( CPUFunction, GPUFunction, N, numThreads ) do { \
    float fMelementsPerSecond; \
    srand(0); \
    bool bSuccess = TestScanWarp<int>( &fMelementsPerSecond, #GPUFunction, CPUFunction, GPUFunction, N, numThreads ); \
    if ( ! bSuccess ) { \
        printf( "%s failed: N=%d, numThreads=%d\n", #GPUFunction, N, numThreads ); \
        exit(1); \
    } \
    if ( fMelementsPerSecond > maxElementsPerSecond ) { \
        maxElementsPerSecond = fMelementsPerSecond; \
    } \
\
} while (0)

    chCommandLineGet( &numInts, "numints", argc, argv );
    printf( "Problem size: %d integers\n", numInts );

{
    float maxElementsPerSecond = 0.0f;
    SCAN_TEST_VECTOR( ScanCPU32<Exclusive>, ScanGPU<Exclusive>, numInts, 256 );
}

#if 0
    for ( int numThreads = 256; numThreads <= maxThreads; numThreads *= 2 ) {
        float maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanCPU32<Exclusive>, ScanGPU<Exclusive>, numInts, numThreads );
        printf( "GPU: %.2f Melements/s\n", maxElementsPerSecond );
        maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanCPU32<Exclusive>, ScanExclusiveGPU_0, numInts, numThreads );
        printf( "GPU: %.2f Melements/s\n", maxElementsPerSecond );
        maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanCPU32<Exclusive>, ScanExclusiveGPU2, numInts, numThreads );
        printf( "GPU2: %.2f Melements/s\n", maxElementsPerSecond );
        maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanCPU32<Exclusive>, ScanGPUShuffle<Exclusive>, numInts, numThreads );
        printf( "Shuffle: %.2f Melements/s\n", maxElementsPerSecond );
    }

    for ( int numThreads = 256; numThreads <= maxThreads; numThreads *= 2 ) {
        float maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanCPU32<Inclusive>, ScanGPU<Inclusive>, numInts, numThreads );
        printf( "GPU: %.2f Melements/s\n", maxElementsPerSecond );
        maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanCPU32<Inclusive>, ScanInclusiveGPU_0, numInts, numThreads );
        printf( "GPU (0): %.2f Melements/s\n", maxElementsPerSecond );
        maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanCPU32<Inclusive>, ScanInclusiveGPU2, numInts, numThreads );
        printf( "GPU2: %.2f Melements/s\n", maxElementsPerSecond );
        maxElementsPerSecond = 0.0f;
        SCAN_TEST_VECTOR( ScanCPU32<Inclusive>, ScanGPUShuffle<Inclusive>, numInts, numThreads );
        printf( "Shuffle: %.2f Melements/s\n", maxElementsPerSecond );
    }
#endif
    return 0;
Error:
    return 1;
}
