#include "hip/hip_runtime.h"
/*
 *
 * stream1Device.cu
 *
 * Microbenchmark to illustrate a bandwidth-limited workload.
 *
 * It separately measures the host->device transfer time, kernel
 * processing time, and device->host transfer time.  Due to low
 * arithmetic density in the saxpy() kernel, the bulk of time
 * is spent transferring data. 
 *
 * Build with: nvcc -I ../chLib stream1Device.cu
 *
 * Copyright (c) 2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <chError.h>
#include <chCommandLine.h>
#include <chTimer.h>

#include <stdio.h>
#include <stdlib.h>

#include "saxpyCPU.h"
#include "saxpyGPU.cuh"

//
// saxpy global function adds x[i]*alpha to each element y[i]
// and writes the result to out[i].
//
// Due to low arithmetic density, this kernel is extremely bandwidth-bound.
//

template<const int n> 
__device__ void
saxpy_unrolled( 
    float *out, 
    const float *px, 
    const float *py, 
    size_t N, 
    float alpha )
{
    float x[n], y[n];
    size_t i;
    for ( i = n*blockIdx.x*blockDim.x+threadIdx.x; 
          i < N-n*blockDim.x*gridDim.x; 
          i += n*blockDim.x*gridDim.x ) {
        for ( int j = 0; j < n; j++ ) {
            size_t index = i+j*blockDim.x;
            x[j] = px[index];
            y[j] = py[index];
        }
        for ( int j = 0; j < n; j++ ) {
            size_t index = i+j*blockDim.x;
            out[index] = alpha*x[j]+y[j];
        }
    }
    // to avoid the (index<N) conditional in the inner loop, 
    // we left off some work at the end
    for ( int j = 0; j < n; j++ ) {
        for ( int j = 0; j < n; j++ ) {
            size_t index = i+j*blockDim.x;
            if ( index<N ) {
                x[j] = px[index];
                y[j] = py[index];
            }
        }
        for ( int j = 0; j < n; j++ ) {
            size_t index = i+j*blockDim.x;
            if ( index<N ) out[index] = alpha*x[j]+y[j];
        }
    }
}

#if 0
__global__ void
saxpyGPU( float *out, const float *px, const float *py, size_t N, float alpha )
{
    saxpy_unrolled<4>( out, px, py, N, alpha );
}
#endif

hipError_t
MeasureTimes( 
    float *msTotal,
    float *msWallClock,
    float *msHtoD, 
    float *msKernel, 
    float *msDtoH, 
    size_t N, 
    float alpha,
    int nBlocks, 
    int nThreads )
{
    hipError_t status;
    chTimerTimestamp chStart, chStop;
    float *dptrOut = 0, *hptrOut = 0;
    float *dptrY = 0, *hptrY = 0;
    float *dptrX = 0, *hptrX = 0;
    hipEvent_t evStart = 0;
    hipEvent_t evHtoD = 0;
    hipEvent_t evKernel = 0;
    hipEvent_t evDtoH = 0;

    hptrOut = new float[N];
    memset( hptrOut, 0, N*sizeof(float) );
    hptrY = new float[N];
    hptrX = new float[N];

    cuda(Malloc( &dptrOut, N*sizeof(float) ) );
    cuda(Memset( dptrOut, 0, N*sizeof(float) ) );

    cuda(Malloc( &dptrY, N*sizeof(float) ) );
    cuda(Memset( dptrY, 0, N*sizeof(float) ) );

    cuda(Malloc( &dptrX, N*sizeof(float) ) );
    cuda(Memset( dptrY, 0, N*sizeof(float) ) );

    cuda(EventCreate( &evStart ) );
    cuda(EventCreate( &evHtoD ) );
    cuda(EventCreate( &evKernel ) );
    cuda(EventCreate( &evDtoH ) );
    for ( size_t i = 0; i < N; i++ ) {
        hptrX[i] = (float) rand() / (float) RAND_MAX;
        hptrY[i] = (float) rand() / (float) RAND_MAX;
    }

    //
    // begin timing
    //

    chTimerGetTime( &chStart );
    cuda(EventRecord( evStart, 0 ) );
    cuda(Memcpy( dptrX, hptrX, N*sizeof(float), hipMemcpyHostToDevice ) );
    cuda(Memcpy( dptrY, hptrY, N*sizeof(float), hipMemcpyHostToDevice ) );
    cuda(EventRecord( evHtoD, 0 ) );
        saxpyGPU<<<nBlocks, nThreads>>>( dptrOut, dptrX, dptrY, N, alpha );
    cuda(EventRecord( evKernel, 0 ) );
    cuda(Memcpy( hptrOut, dptrOut, N*sizeof(float), hipMemcpyDeviceToHost ) );
    cuda(EventRecord( evDtoH, 0 ) );
    cuda(DeviceSynchronize() );

    //
    // end timing
    //

    chTimerGetTime( &chStop );
    *msWallClock = 1000.0f*chTimerElapsedTime( &chStart, &chStop );
    for ( size_t i = 0; i < N; i++ ) {
        if ( fabsf( hptrOut[i] - (alpha*hptrX[i]+hptrY[i]) ) > 1e-5f ) {
            status = hipErrorUnknown;
            goto Error;
        }
    }
    cuda(EventElapsedTime( msHtoD, evStart, evHtoD ) );
    cuda(EventElapsedTime( msKernel, evHtoD, evKernel ) );
    cuda(EventElapsedTime( msDtoH, evKernel, evDtoH ) );
    cuda(EventElapsedTime( msTotal, evStart, evDtoH ) );
Error:
    hipEventDestroy( evDtoH );
    hipEventDestroy( evKernel );
    hipEventDestroy( evHtoD );
    hipEventDestroy( evStart );
    hipFree( dptrOut );
    hipFree( dptrX );
    hipFree( dptrY );
    delete[] hptrOut;
    delete[] hptrX;
    delete[] hptrY;
    return status;
}

double
Bandwidth( float ms, double NumBytes )
{
    return NumBytes / (1000.0*ms);
}

int
main( int argc, char *argv[] )
{
    int device = 0;
    hipError_t status;
    int N_Mfloats = 128;
    size_t N;
    int nBlocks = 1500;
    int nThreads = 256;
    float alpha = 2.0f;

    chCommandLineGet( &device, "device", argc, argv );
    chCommandLineGet( &nBlocks, "nBlocks", argc, argv );
    chCommandLineGet( &nThreads, "nThreads", argc, argv );
    chCommandLineGet( &N_Mfloats, "N", argc, argv );
    printf( "Measuring times with %dM floats", N_Mfloats );
    if ( N_Mfloats==128 ) {
        printf( " (use --N to specify number of Mfloats)");
    }
    printf( "\n" );

    N = 1048576*N_Mfloats;

    cuda(SetDevice( device ) );
    cuda(SetDeviceFlags( hipDeviceMapHost ) );
    {
        float msTotal, msWallClock, msHtoD, msKernel, msDtoH;
        CUDART_CHECK( MeasureTimes( &msTotal, &msWallClock, &msHtoD, &msKernel, &msDtoH, N, alpha, nBlocks, nThreads ) );
        printf( "Memcpy( host->device ): %.2f ms (%.2f MB/s)\n", msHtoD, Bandwidth( msHtoD, 2*N*sizeof(float) ) );
        printf( "Kernel processing     : %.2f ms (%.2f MB/s)\n", msKernel, Bandwidth( msKernel, 3*N*sizeof(float) ) );
        printf( "Memcpy (device->host ): %.2f ms (%.2f MB/s)\n\n", msDtoH, Bandwidth( msDtoH, N*sizeof(float) ) );
        printf( "Total time (wall clock): %.2f ms (%.2f MB/s)\n", msWallClock, Bandwidth( msWallClock, 3*N*sizeof(float) ) );
    }

Error:
    if ( status == hipErrorMemoryAllocation ) {
        printf( "Memory allocation failed\n" );
    }
    else if ( hipSuccess != status ) {
        printf( "Failed\n" );
    }
    return hipSuccess != status;
}
