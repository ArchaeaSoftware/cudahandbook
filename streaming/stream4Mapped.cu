#include "hip/hip_runtime.h"
/*
 *
 * stream4Mapped.cu
 *
 * Formulation of stream1Async.cu that uses mapped pinned memory to
 * hold the input and output data.  Since the kernel can use mapped
 * pinned memory to initiate DMA transfers across the bus, this
 * version is simpler (no hipMemcpy() calls) and just as fast.
 *
 * Build with: nvcc -I ../chLib stream4Mapped.cu
 *
 * Copyright (c) 2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <chError.h>
#include <chCommandLine.h>
#include <chTimer.h>

#include <stdio.h>
#include <stdlib.h>

#include "saxpyCPU.h"
#include "saxpyGPU.cuh"

hipError_t
MeasureTimes( 
    float *msTotal,
    float *msWallClock,
    size_t N, 
    float alpha,
    int nBlocks, 
    int nThreads )
{
    hipError_t status;
    chTimerTimestamp chStart, chStop;
    float *dptrOut = 0, *hptrOut = 0;
    float *dptrY = 0, *hptrY = 0;
    float *dptrX = 0, *hptrX = 0;
    hipEvent_t evStart = 0;
    hipEvent_t evStop = 0;

    cuda(HostAlloc( &hptrOut, N*sizeof(float), hipHostMallocMapped ) );
    cuda(HostGetDevicePointer( &dptrOut, hptrOut, 0 ) );
    memset( hptrOut, 0, N*sizeof(float) );

    cuda(HostAlloc( &hptrY, N*sizeof(float), hipHostMallocMapped ) );
    cuda(HostGetDevicePointer( &dptrY, hptrY, 0 ) );
    cuda(HostAlloc( &hptrX, N*sizeof(float), hipHostMallocMapped ) );
    cuda(HostGetDevicePointer( &dptrX, hptrX, 0 ) );

    cuda(EventCreate( &evStart ) );
    cuda(EventCreate( &evStop ) );
    for ( size_t i = 0; i < N; i++ ) {
        hptrX[i] = (float) ((double) rand() / RAND_MAX);
        hptrY[i] = (float) ((double) rand() / RAND_MAX);
    }

    //
    // begin timing
    //

    chTimerGetTime( &chStart );
    cuda(EventRecord( evStart, 0 ) );
        saxpyGPU<<<nBlocks, nThreads>>>( dptrOut, dptrX, dptrY, N, alpha );
    cuda(EventRecord( evStop, 0 ) );
    cuda(DeviceSynchronize() );

    //
    // end timing
    //

    chTimerGetTime( &chStop );
    *msWallClock = 1000.0f*chTimerElapsedTime( &chStart, &chStop );
    for ( size_t i = 0; i < N; i++ ) {
        if ( fabsf( hptrOut[i] - (alpha*hptrX[i]+hptrY[i]) ) > 1e-5f ) {
            status = hipErrorUnknown;
            goto Error;
        }
    }
    cuda(EventElapsedTime( msTotal, evStart, evStop ) );
Error:
    hipEventDestroy( evStop );
    hipEventDestroy( evStart );
    cudaHostFree( hptrOut );
    cudaHostFree( hptrX );
    cudaHostFree( hptrY );
    return status;
}

double
Bandwidth( float ms, double NumBytes )
{
    return NumBytes / (1000.0*ms);
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    int N_Mfloats = 128;
    size_t N;
    int nBlocks = 1500;
    int nThreads = 256;
    float alpha = 2.0f;

    chCommandLineGet( &nBlocks, "nBlocks", argc, argv );
    chCommandLineGet( &nThreads, "nThreads", argc, argv );
    chCommandLineGet( &N_Mfloats, "N", argc, argv );
    printf( "Measuring times with %dM floats", N_Mfloats );
    if ( N_Mfloats==128 ) {
        printf( " (use --N to specify number of Mfloats)");
    }
    printf( "\n" );

    N = 1048576*N_Mfloats;

    cuda(SetDeviceFlags( hipDeviceMapHost ) );
    {
        float msTotalGPU, msWallClock;
        CUDART_CHECK( MeasureTimes( &msTotalGPU, &msWallClock, N, alpha, nBlocks, nThreads ) );
        printf( "Total time (GPU event):  %.2f ms (%.2f MB/s)\n", msTotalGPU, Bandwidth( msTotalGPU, 3*N*sizeof(float) ) );
        printf( "Total time (wall clock): %.2f ms (%.2f MB/s)\n", msWallClock, Bandwidth( msWallClock, 3*N*sizeof(float) ) );
    }

Error:
    if ( status == hipErrorOutOfMemory ) {
        printf( "Memory allocation failed\n" );
    }
    else if ( hipSuccess != status ) {
        printf( "Failed\n" );
    }
    return hipSuccess != status;
}
