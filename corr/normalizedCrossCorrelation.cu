#include "hip/hip_runtime.h"
/*
 *
 * normalizedCrossCorrelation.cu
 *
 * Microbenchmark for normalized cross correlation, a template-
 * matching algorithm for computer vision.
 *
 * Build with: nvcc -I ../chLib <options> normalizedCrossCorrelation.cu ..\chLib\pgm.cu
 *
 * Make sure to include pgm.cu for the image file I/O support.
 *
 * To avoid warnings about double precision support, specify the
 * target gpu-architecture, e.g.:
 * nvcc --gpu-architecture sm_13 -I ../chLib <options> normalizedCrossCorrelation.cu pgm.cu
 *
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <chError.h>
#include <chCommandLine.h>
#include <chAssert.h>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <assert.h>

#include "pgm.h"

texture<unsigned char, 2> texImage;
texture<unsigned char, 2> texTemplate;

const int maxTemplatePixels = 3072;
__constant__ int g_xOffset[maxTemplatePixels];
__constant__ int g_yOffset[maxTemplatePixels];
__constant__ unsigned char g_Tpix[maxTemplatePixels];
__constant__ float g_cPixels, g_SumT, g_fDenomExp;
unsigned int g_cpuSumT, g_cpuSumTSq;

const float fThreshold = 1e-3f;

#define INTCEIL(a,b) ( ((a)+(b)-1) / (b) )

__device__ __host__ inline float
CorrelationValue( float SumI, float SumISq, float SumIT, float SumT, float cPixels, float fDenomExp )
{
    float Numerator = cPixels*SumIT - SumI*SumT;
    float Denominator = rsqrtf( (cPixels*SumISq - SumI*SumI)*fDenomExp );
    return Numerator * Denominator;
}

#include "corrTexTexSums.cuh"
#include "corrTexTex.cuh"

#include "corrTexConstantSums.cuh"
#include "corrTexConstant.cuh"

extern __shared__ unsigned char LocalBlock[];

#include "corrSharedSMSums.cuh"
#include "corrSharedSM.cuh"

#include "corrSharedSums.cuh"
#include "corrShared.cuh"

#include "corrShared4Sums.cuh"
#include "corrShared4.cuh"

int poffsetx[maxTemplatePixels];
int poffsety[maxTemplatePixels];

hipError_t
CopyToTemplate( 
      unsigned char *img, size_t imgPitch, 
      int xTemplate, int yTemplate,
      int wTemplate, int hTemplate,
      int OffsetX, int OffsetY
)
{
    hipError_t status;
    unsigned char pixels[maxTemplatePixels];

    int inx = 0;
    int SumT = 0;
    int SumTSq = 0;
    int cPixels = wTemplate*hTemplate;
    size_t sizeOffsets = cPixels*sizeof(int);
    float fSumT, fDenomExp, fcPixels;

    cuda(Memcpy2D( 
        pixels, wTemplate,
        img+yTemplate*imgPitch+xTemplate, imgPitch,
        wTemplate, hTemplate,
        hipMemcpyDeviceToHost ) );

    cuda(MemcpyToSymbol( g_Tpix, pixels, cPixels ) );

    for ( int i = OffsetY; i < OffsetY+hTemplate; i++ ) {
        for ( int j = OffsetX; j < OffsetX+wTemplate; j++) {
            SumT += pixels[inx];
            SumTSq += pixels[inx]*pixels[inx];
            poffsetx[inx] = j;
            poffsety[inx] = i;
            inx += 1;
        }
    }
    g_cpuSumT = SumT;
    g_cpuSumTSq = SumTSq;

    cuda(MemcpyToSymbol(g_xOffset, poffsetx, sizeOffsets) );
    cuda(MemcpyToSymbol(g_yOffset, poffsety, sizeOffsets) );

    fSumT = (float) SumT;
    cuda(MemcpyToSymbol(g_SumT, &fSumT, sizeof(float)) );

    fDenomExp = float( (double)cPixels*SumTSq - (double) SumT*SumT);
    cuda(MemcpyToSymbol(g_fDenomExp, &fDenomExp, sizeof(float)) );

    fcPixels = (float) cPixels;
    cuda(MemcpyToSymbol(g_cPixels, &fcPixels, sizeof(float)) );
Error:
    return status;
}

int
bCompareCorrValues( const float *pBase0, 
                    const float *pBase1, 
                    int w, int h )
{
    for ( int j = 0; j < h; j++ ) {

        float *pf0 = (float *) ((char *) pBase0+j*w*sizeof(float));
        float *pf1 = (float *) ((char *) pBase1+j*w*sizeof(float));

        for ( int i = 0; i < w; i++ ) {
            if ( fabsf(pf0[i]-pf1[i]) > fThreshold ) { 
                printf( "Mismatch pf0[%d] = %.5f, pf1[%d] = %.5f\n", i, pf0[i], i, pf1[i] ); 
                fflush( stdout );
                //CH_ASSERT(0);
                return 1;
            }
        }
    }
    return 0;
}

int
bCompareSums( const int *pBaseI0, const int *pBaseISq0, const int *pBaseIT0,
              const int *pBaseI1, const int *pBaseISq1, const int *pBaseIT1,
              int w, int h )
{
    for ( int j = 0; j < h; j++ ) {

        const int *pi0 = (const int *) ((char *) pBaseI0+j*w*sizeof(int));
        const int *pi1 = (const int *) ((char *) pBaseI1+j*w*sizeof(int));

        const int *pisq0 = (const int *) ((char *) pBaseISq0+j*w*sizeof(int));
        const int *pisq1 = (const int *) ((char *) pBaseISq1+j*w*sizeof(int));

        const int *pit0 = (const int *) ((char *) pBaseIT0+j*w*sizeof(int));
        const int *pit1 = (const int *) ((char *) pBaseIT1+j*w*sizeof(int));
        for ( int i = 0; i < w; i++ ) {
            if ( pi0[i] != pi1[i] ||
                 pisq0[i] != pisq1[i] ||
                 pit0[i] != pit1[i] ) { 
                printf( "Mismatch pi[%d] = %d, reference = %d\n", i, pi0[i], pi1[i] ); 
                printf( "Mismatch pisq[%d] = %d, reference = %d\n", i, pisq0[i], pisq1[i] );
                printf( "Mismatch pit[%d] = %d, reference = %d\n", i, pit0[i], pit1[i] );
                fflush( stdout );
                //CH_ASSERT(0);
                return 1;
            }
        }
    }
    return 0;
}

unsigned char
ReadPixel( unsigned char *base, int pitch, int w, int h, int x, int y )
{
    if ( x < 0 ) x = 0;
    if ( x >= w ) x = w-1;
    if ( y < 0 ) y = 0;
    if ( y >= h ) y = h-1;
    return base[y*pitch+x];
}

void 
corrCPU( float *pCorr, 
         int *_pI, int *_pISq, int *_pIT,
         size_t CorrPitch, 
         int cPixels,
         int xTemplate, int yTemplate,
         int w, int h,
         unsigned char *img, int imgPitch,
         unsigned char *tmp, int tmpPitch )
{
    for ( int row = 0; row < h; row += 1 ) {
        float *pOut = (float *) (((char *) pCorr)+row*CorrPitch);
        int *pI = (int *) (((char *) _pI)+row*CorrPitch);
        int *pISq = (int *) (((char *) _pISq)+row*CorrPitch);
        int *pIT = (int *) (((char *) _pIT)+row*CorrPitch);
        for ( int col = 0; col < w; col += 1 ) {
            int SumI = 0;
            int SumT = 0;
            int SumISq = 0;
            int SumTSq = 0;
            int SumIT = 0;
            for ( int j = 0; j < cPixels; j++ ) {
                unsigned char I = ReadPixel( img, imgPitch, w, h, col+poffsetx[j], row+poffsety[j] );
                unsigned char T = ReadPixel( tmp, tmpPitch, w, h, xTemplate+poffsetx[j], yTemplate+poffsety[j] );
                SumI += I;
                SumT += T;
                SumISq += I*I;
                SumTSq += T*T;
                SumIT += I*T;
            }
            float fDenomExp = float((double) cPixels*SumTSq - (double) SumT*SumT);
            pI[col] = SumI;
            pISq[col] = SumISq;
            pIT[col] = SumIT;
            pOut[col] = CorrelationValue( (float) SumI, (float) SumISq, (float) SumIT, (float) SumT, (float) cPixels, fDenomExp );
        }
    }
}

bool
TestCorrelation( 
    double *pixelsPerSecond,         // passbacks to report performance
    double *templatePixelsPerSecond, // 
    int xOffset, int yOffset,  // offset into image
    int w, int h,              // width and height of output
    const float *hrefCorr,     // host reference data
    const int *hrefSumI,
    const int *hrefSumISq, 
    const int *hrefSumIT,
    int xTemplate, int yTemplate, // reference point in template image
    int wTemplate, int hTemplate,
    int wTile,                 // width of image tile
    int sharedPitch, int sharedMem,
    dim3 threads, dim3 blocks,
    void (*pfnCorrelationSums)( 
        float *dCorr, int CorrPitch,
        int *dSumI, int *dSumISq, int *dSumIT,
        int wTile,
        int wTemplate, int hTemplate,
        float cPixels,
        float fDenomExp,
        int sharedPitch,
        int xOffset, int yOffset,
        int xTemplate, int yTemplate,
        int xUL, int yUL, int w, int h,
        dim3 threads, dim3 blocks,
        int sharedMem ),
    void (*pfnCorrelation)( 
        float *dCorr, int CorrPitch,
        int wTile,
        int wTemplate, int hTemplate,
        float cPixels,
        float fDenomExp,
        int sharedPitch,
        int xOffset, int yOffset,
        int xTemplate, int yTemplate,
        int xUL, int yUL, int w, int h,
        dim3 threads, dim3 blocks,
        int sharedMem ),
    bool bPrintNeighborhood = false,
    int cIterations = 1,
    const char *outputFilename = NULL
)
{
    hipError_t status;
    bool ret = false;
    size_t CorrPitch;

    float cPixels = (float) wTemplate*hTemplate;
    float fDenomExp = float((double) cPixels*g_cpuSumTSq - (double) g_cpuSumT*g_cpuSumT);

    float *hCorr = NULL, *dCorr = NULL;
    int *hSumI = NULL, *dSumI = NULL;
    int *hSumISq = NULL, *dSumISq = NULL;
    int *hSumIT = NULL, *dSumIT = NULL;

    hipEvent_t start = 0, stop = 0;

    hCorr = (float *) malloc( w*sizeof(float)*h );
    hSumI = (int *) malloc( w*sizeof(int)*h );
    hSumISq = (int *) malloc( w*sizeof(int)*h );
    hSumIT = (int *) malloc( w*sizeof(int)*h );
    if ( NULL == hCorr || NULL == hSumI || NULL == hSumISq || NULL == hSumIT )
        goto Error;

    cuda(MallocPitch( (void **) &dCorr, &CorrPitch, w*sizeof(float), h ) );
    cuda(MallocPitch( (void **) &dSumI, &CorrPitch, w*sizeof(int), h ) );
    cuda(MallocPitch( (void **) &dSumISq, &CorrPitch, w*sizeof(int), h ) );
    cuda(MallocPitch( (void **) &dSumIT, &CorrPitch, w*sizeof(int), h ) );

    cuda(Memset( dCorr, 0, CorrPitch*h ) );
    cuda(Memset( dSumI, 0, CorrPitch*h ) );
    cuda(Memset( dSumISq, 0, CorrPitch*h ) );
    cuda(Memset( dSumIT, 0, CorrPitch*h ) );

    cuda(EventCreate( &start, 0 ) );
    cuda(EventCreate( &stop, 0 ) );

    pfnCorrelationSums(
        dCorr, CorrPitch,
        dSumI, dSumISq, dSumIT,
        wTile,
        wTemplate, hTemplate, 
        cPixels, fDenomExp, 
        sharedPitch, 
        xOffset, yOffset, 
        xTemplate, yTemplate, 
        0, 0, w, h,
        threads, blocks, sharedMem );

    cuda(Memcpy2D( hSumI, w*sizeof(int), dSumI, CorrPitch, w*sizeof(int), h, hipMemcpyDeviceToHost ) );
    cuda(Memcpy2D( hSumISq, w*sizeof(int), dSumISq, CorrPitch, w*sizeof(int), h, hipMemcpyDeviceToHost ) );
    cuda(Memcpy2D( hSumIT, w*sizeof(int), dSumIT, CorrPitch, w*sizeof(int), h, hipMemcpyDeviceToHost ) );

    if ( bCompareSums( hSumI, hSumISq, hSumIT,
                       hrefSumI, hrefSumISq, hrefSumIT,
                       w, h ) ) {
        //CH_ASSERT(0);
        printf( "Sums miscompare\n" );
        goto Error;
    }

    cuda(Memcpy2D( hCorr, w*sizeof(float), dCorr, CorrPitch, w*sizeof(float), h, hipMemcpyDeviceToHost ) );

    if ( bCompareCorrValues( hrefCorr, hCorr, w, h ) ) {
        //CH_ASSERT(0);
        printf( "Correlation coefficients generated by sums kernel mismatch\n" );
        return 1;
    }

    cuda(Memset2D( dCorr, CorrPitch, 0, w*sizeof(float), h ) );
    cuda(DeviceSynchronize() );
    cuda(EventRecord( start, 0 ) );

    for ( int i = 0; i < cIterations; i++ ) {
        pfnCorrelation( 
            dCorr, CorrPitch, 
            wTile, 
            wTemplate, hTemplate, 
            cPixels, fDenomExp, 
            sharedPitch, 
            xOffset, yOffset,
            xTemplate, yTemplate, 
            0, 0, w, h, 
            threads, blocks, sharedMem );
    }

    cuda(EventRecord( stop, 0 ) );
    cuda(Memcpy2D( hCorr, w*sizeof(float), dCorr, CorrPitch, w*sizeof(float), h, hipMemcpyDeviceToHost ) );

    if ( bCompareCorrValues( hrefCorr, hCorr, w, h ) ) {
        CH_ASSERT(0);
        printf( "Correlation coefficients generated by coefficient-only kernel mismatch\n" );
        return 1;
    }

    {
        float ms;
        cuda(EventElapsedTime( &ms, start, stop ) );
        *pixelsPerSecond = (double) w*h*cIterations*1000.0 / ms;
        *templatePixelsPerSecond = *pixelsPerSecond*wTemplate*hTemplate;
    }

    if ( bPrintNeighborhood ) {
        printf( "\nNeighborhood around template:\n" );
        for ( int VertOffset = -4; VertOffset <= 4; VertOffset++ ) {
            const float *py = hrefCorr+w*(VertOffset+yTemplate);
            for ( int HorzOffset = -4; HorzOffset <= 4; HorzOffset++ ) {
                printf( "%6.2f", py[xTemplate+HorzOffset] );
            }
            printf("\n");
        }
    }

    if ( outputFilename ) {
        unsigned char *correlationValues = (unsigned char *) malloc( w*h );
        if ( ! correlationValues ) {
            status = hipErrorOutOfMemory;
            goto Error;
        }
        for ( int row = 0; row < h; row++ ) {
            for ( int col = 0; col < w; col++ ) {
                int index = row*w+col;
                float value = hCorr[index] < 0.0f ? 0.0f : logf( 1.0f+hCorr[index] )/logf(2.0f);
                if ( value < 0.5f ) value = 0.0f;
                value = 2.0f * (value - 0.5f);
                correlationValues[index] = (unsigned char) (255.0f*value+0.5f);
            }
        }
        if ( 0 != pgmSave( outputFilename, correlationValues, w, h ) ) {
            status = hipErrorUnknown;
            goto Error;
        }
        free( correlationValues );
    }

    ret = true;

Error:
    hipEventDestroy( start );
    hipEventDestroy( stop );
    free( hCorr );
    free( hSumI );
    free( hSumISq );
    free( hSumIT );
    if ( dCorr ) hipFree( dCorr );
    if ( dSumI ) hipFree( dSumI );
    if ( dSumI ) hipFree( dSumISq );
    if ( dSumI ) hipFree( dSumIT );
    return ret;
}

int
main(int argc, char *argv[])
{
    int ret = 1;
    hipError_t status;

    unsigned char *hidata = NULL;
    unsigned char *didata = NULL;
    float *hoCorrCPU = NULL;
    
    int *hoCorrCPUI = NULL;
    int *hoCorrCPUISq = NULL;
    int *hoCorrCPUIT = NULL;
    unsigned int HostPitch, DevicePitch;
    int w, h;

    int wTemplate = 52;
    int hTemplate = 52;
    int xOffset, yOffset;

    int xTemplate = 210;
    int yTemplate = 148;

    int wTile;
    dim3 threads;
    dim3 blocks;

    int sharedPitch;
    int sharedMem;
    char defaultInputFilename[] = "coins.pgm";
    char *inputFilename = defaultInputFilename;
    char *outputFilename = NULL;

    hipArray *pArrayImage = NULL;
    hipArray *pArrayTemplate = NULL;
    hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();

    if ( chCommandLineGetBool( "help", argc, argv ) ) {
        printf( "Usage:\n" );
        printf( "    --input <filename>: specify input filename (must be PGM)\n" );
        printf( "    --output <filename>: Write PGM of correlation values (0..255) to <filename>.\n" );
        printf( "    --padWidth <value>: pad input image width to specified value\n" );
        printf( "    --padHeight <value>: pad input image height to specified value\n" );
        printf( "    --xTemplate <value>: X coordinate of upper left corner of template\n" );
        printf( "    --yTemplate <value>: Y coordinate of upper left corner of template\n" );
        printf( "    --wTemplate <value>: Width of template\n" );
        printf( "    --hTemplate <value>: Height of template\n" );
        printf( "\nDefault values are coins.pgm, no output file or padding, and template of the dime in the\n" );
        printf("lower right corner of coins.pgm: xTemplate=210, yTemplate=148, wTemplate=hTemplate=52\n" );

        return 0;
    }

    cuda(SetDeviceFlags( hipDeviceMapHost ) );
    cuda(DeviceSetCacheConfig( hipFuncCachePreferShared ) );

    if ( chCommandLineGet( &inputFilename, "input", argc, argv ) ) {
        printf( "Reading from image file %s\n", inputFilename );
    }
    chCommandLineGet( &outputFilename, "output", argc, argv );
    {
        int padWidth = 0;
        int padHeight = 0;
        if ( chCommandLineGet( &padWidth, "padWidth", argc, argv ) ) {
            if ( ! chCommandLineGet( &padHeight, "padHeight", argc, argv ) ) {
                printf( "Must specify both --padWidth and --padHeight\n" );
                goto Error;
            }
        }
        else {
            if ( chCommandLineGet( &padHeight, "padHeight", argc, argv ) ) {
                printf( "Must specify both --padWidth and --padHeight\n" );
                goto Error;
            }
        }
        if ( pgmLoad(inputFilename, &hidata, &HostPitch, &didata, &DevicePitch, &w, &h, padWidth, padHeight) )
            goto Error;
    }
    chCommandLineGet( &xTemplate, "xTemplate", argc, argv );
    chCommandLineGet( &yTemplate, "yTemplate", argc, argv );
    chCommandLineGet( &wTemplate, "wTemplate", argc, argv );
    chCommandLineGet( &hTemplate, "hTemplate", argc, argv );

    xOffset = -wTemplate/2;
    yOffset = -wTemplate/2;

    hoCorrCPU = (float *) malloc(w*h*sizeof(float)); if ( ! hoCorrCPU ) return 1;
    hoCorrCPUI = (int *) malloc(w*h*sizeof(int)); if ( ! hoCorrCPUI ) return 1;
    hoCorrCPUISq = (int *) malloc(w*h*sizeof(int)); if ( ! hoCorrCPUISq ) return 1;
    hoCorrCPUIT = (int *) malloc(w*h*sizeof(int)); if ( ! hoCorrCPUIT ) return 1;
    if ( NULL == hoCorrCPU ||
         NULL == hoCorrCPUI ||
         NULL == hoCorrCPUISq ||
         NULL == hoCorrCPUIT )
        goto Error;

    cuda(MallocArray( &pArrayImage, &desc, w, h ) );
    cuda(MallocArray( &pArrayTemplate, &desc, w, h ) );
    cuda(MemcpyToArray( pArrayImage, 0, 0, hidata, w*h, hipMemcpyHostToDevice ) );
        
    cuda(Memcpy2DArrayToArray( pArrayTemplate, 0, 0, pArrayImage, 0, 0, w, h, hipMemcpyDeviceToDevice ) );
    
    cuda(BindTextureToArray( texImage, pArrayImage ) );
    cuda(BindTextureToArray( texTemplate, pArrayTemplate ) );

    CopyToTemplate( didata, DevicePitch, 
                    xTemplate, yTemplate, 
                    wTemplate, hTemplate,
                    xOffset, yOffset );

    corrCPU( hoCorrCPU, hoCorrCPUI, hoCorrCPUISq, hoCorrCPUIT, 
        w*sizeof(float), wTemplate*hTemplate, xTemplate-xOffset, yTemplate-yOffset, w, h, 
        hidata, HostPitch, hidata, HostPitch );

    // height of thread block must be >= hTemplate
    wTile = 32;
    threads = dim3(32,8);
    blocks = dim3(w/wTile+(0!=w%wTile),h/threads.y+(0!=h%threads.y));

    sharedPitch = ~63&(wTile+wTemplate+63);
    sharedMem = sharedPitch*(threads.y+hTemplate);

#define TEST_VECTOR( baseName, bPrintNeighborhood, cIterations, outfile ) \
    { \
        double pixelsPerSecond; \
        double templatePixelsPerSecond; \
        if ( ! TestCorrelation( &pixelsPerSecond, \
            &templatePixelsPerSecond, \
            xOffset, yOffset, \
            w, h,  \
            hoCorrCPU, \
            hoCorrCPUI, \
            hoCorrCPUISq, \
            hoCorrCPUIT, \
            xTemplate-xOffset, yTemplate-yOffset, \
            wTemplate, hTemplate, \
            wTile, sharedPitch, sharedMem, \
            threads, blocks,  \
            baseName##Sums,  \
            baseName, \
            bPrintNeighborhood, cIterations, outfile ) ) { \
            printf( "Error\n" ); \
        } \
        printf( "%s: %.2f Mpix/s\t%.2fGtpix/s\n", \
            #baseName, pixelsPerSecond/1e6, templatePixelsPerSecond/1e9 ); \
    }

    TEST_VECTOR( corrShared, false, 100, NULL );

    // height of thread block must be >= hTemplate
    wTile = 32;
    threads = dim3(32,8);
    blocks = dim3(w/wTile+(0!=w%wTile),h/threads.y+(0!=h%threads.y));

    sharedPitch = ~63&(((wTile+wTemplate)+63));
    sharedMem = sharedPitch*(threads.y+hTemplate);

    TEST_VECTOR( corrSharedSM, false, 100, NULL );

    TEST_VECTOR( corrShared4, false, 100, NULL );

    // set up blocking parameters for 2D tex-constant formulation
    threads.x = 32; threads.y = 16; threads.z = 1;
    blocks.x = INTCEIL(w,threads.x); blocks.y = INTCEIL(h,threads.y); blocks.z = 1;
    TEST_VECTOR( corrTexConstant, false, 100, NULL );

    if ( outputFilename ) {
        printf( "Writing graymap of correlation values to %s\n", outputFilename );
    }

    // set up blocking parameters for 2D tex-tex formulation
    threads.x = 16; threads.y = 8; threads.z = 1;
    blocks.x = INTCEIL(w,threads.x); blocks.y = INTCEIL(h,threads.y); blocks.z = 1;
    TEST_VECTOR( corrTexTex, false, 100, outputFilename );

    ret = 0;
Error:
    free( hoCorrCPU );
    free( hoCorrCPUI );
    free( hoCorrCPUISq );
    free( hoCorrCPUIT );

    free( hidata );

    hipFree(didata); 

    hipFreeArray(pArrayImage);
    hipFreeArray(pArrayTemplate);
   
    return ret;

}
