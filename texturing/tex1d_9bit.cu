#include "hip/hip_runtime.h"
/*
 *
 * tex1d_9bit.cu
 *
 * Microdemo to illustrate the 9-bit precision limitation of the
 * linear interpolation performed by the texture units.
 *
 * Build with: nvcc -I ../chLib <options> tex1d_9bit.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include <chError.h>

texture<float, 1> tex;

extern "C" __global__ void
TexReadout( float2 *out, size_t N, float base, float increment )
{
    for ( size_t i = blockIdx.x*blockDim.x + threadIdx.x; 
          i < N; 
          i += gridDim.x*blockDim.x )
    {
        float x = base + (float) i * increment;
        out[i].x = x;
        out[i].y = tex1D( tex, x );
    }
}

template<class T>
void
CreateAndPrintTex( T *initTex, size_t texN, size_t outN, 
    float base, float increment, float expectedBase, float expectedIncrement,
    bool bEmulateGPU = false,
    hipTextureFilterMode filterMode = hipFilterModeLinear, 
    hipTextureAddressMode addressMode = hipAddressModeClamp )
{
    T *texContents = 0;
    hipArray *texArray = 0;

    float2 *outHost = 0, *outDevice = 0;
    hipError_t status;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();

    // use caller-provided array, if any, to initialize texture
    if ( initTex ) {
        texContents = initTex;
    }
    else {
        // default is to initialize with identity elements
        texContents = (T *) malloc( texN*sizeof(T) );
        if ( ! texContents )
            goto Error;
        for ( int i = 0; i < texN; i++ ) {
            texContents[i] = (T) i;
        }
    }

    cuda(MallocArray(&texArray, &channelDesc, texN));

    cuda(HostAlloc( (void **) &outHost, outN*sizeof(float2), hipHostMallocMapped));
    cuda(HostGetDevicePointer( (void **) &outDevice, outHost, 0 ));

    cuda(MemcpyToArray( texArray, 0, 0, texContents, 
                                  texN*sizeof(T), hipMemcpyHostToDevice));
    cuda(BindTextureToArray(tex, texArray));

    tex.filterMode = filterMode;
    tex.addressMode[0] = addressMode;
    cuda(HostGetDevicePointer(&outDevice, outHost, 0));
    TexReadout<<<2,384>>>( outDevice, outN, base, increment );
    cuda(DeviceSynchronize());

    printf( "X\tY\tActual Value\tExpected Value\tDiff\n" );
    for ( int i = 0; i < outN; i++ ) {
        T expected;
        if ( bEmulateGPU ) {
            float x = base+(float)i*increment - 0.5f;
            float frac = x - (float) (int) x;
            {
                int frac256 = (int) (frac*256.0f+0.5f);
                frac = frac256/256.0f;
            }
            int index = (int) x;
            expected = (1.0f-frac)*initTex[index] + 
                              frac*initTex[index+1];
        }
        else {
            expected = expectedBase + (float) i*expectedIncrement;
        }
        float diff = fabsf( outHost[i].y - expected );
        printf( "%.2f\t%.2f\t", outHost[i].x, outHost[i].y );
        printf( "%08x\t", *(int *) (&outHost[i].y) );
        printf( "%08x\t", *(int *) (&expected) );
        printf( "%E\n", diff );
    }
    printf( "\n" );

Error:
    if ( ! initTex ) free( texContents );
    if ( texArray ) hipFreeArray( texArray );
    if ( outHost ) hipHostFree( outHost );
}

hipError_t
init()
{
    hipError_t status;
    cu(Init(0));
Error:;
    return status;
}

int
main( int argc, char *argv[] )
{
    int ret = 1;
    hipError_t status;

    init();

    cuda(SetDeviceFlags(hipDeviceMapHost));

    {
        float texData[10];
        for ( int i = 0; i < 10; i++ ) {
            texData[i] = (float) i / 10.0f;
        }

        CreateAndPrintTex<float>( texData, 10, 4, 1.5f, 0.25f, 0.1f, 0.025f );
        CreateAndPrintTex<float>( texData, 10, 4, 1.5f, 0.1f, 0.1f, 0.01f );

        CreateAndPrintTex<float>( texData, 10, 4, 1.5f, 0.25f, 0.1f, 0.025f, true );
        CreateAndPrintTex<float>( texData, 10, 4, 1.5f, 0.1f, 0.1f, 0.01f, true );
    }
    ret = 0;
Error:
    return ret;
}
