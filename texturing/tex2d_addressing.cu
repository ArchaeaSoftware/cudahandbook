#include "hip/hip_runtime.h"
/*
 *
 * tex2d_addressing.cu
 *
 * Microdemo for 2D texturing addressing modes.
 *
 * Build with: nvcc -I ../chLib <options> tex2d_addressing.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <float.h>
#include <assert.h>

#include <chError.h>

texture<float2, 2, hipReadModeElementType> tex;

extern "C" __global__ void
TexReadout( 
    float4 *out, 
    size_t Width, 
    size_t Pitch, 
    size_t Height, 
    float2 base, 
    float2 increment )
{
    for ( int row = blockIdx.y*blockDim.y + threadIdx.y;
              row < Height;
              row += blockDim.y*gridDim.y )
    {
        float4 *outrow = (float4 *) ((char *) out+row*Pitch);
        for ( int col = blockIdx.x*blockDim.x + threadIdx.x;
                  col < Width;
                  col += blockDim.x*gridDim.x )
        {
            float4 value;
            float2 texvalue;
            value.x = base.x+(float)col*increment.x;
            value.y = base.y+(float)row*increment.y;

            texvalue = tex2D( tex, value.x, value.y);
            value.z = texvalue.x;
            value.w = texvalue.y;
            outrow[col] = value;
        }
    }
}

template<class T>
void
CreateAndPrintTex( 
    T *initTex, 
    size_t inWidth, size_t inHeight, 
    size_t outWidth, size_t outHeight,
    float2 base, float2 increment, 
    hipTextureFilterMode filterMode, 
    hipTextureAddressMode addressModeX,
    hipTextureAddressMode addressModeY )
{
    T *texContents = 0;
    hipArray *texArray = 0;
    float4 *outHost = 0, *outDevice = 0;
    hipError_t status;
    size_t outPitch;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();
    dim3 blocks, threads;

    // use caller-provided array, if any, to initialize texture
    if ( initTex ) {
        texContents = initTex;
    }
    else {
        // default is to initialize with identity elements
        texContents = (T *) malloc( inWidth*inHeight*sizeof(T) );
        if ( ! texContents )
            goto Error;
        for ( int row = 0; row < inHeight; row++ ) {
            T *rowptr = texContents + row*inWidth;
            for ( int col = 0; col < outHeight; col++ ) {
                T value;
                value.x = (float) col;
                value.y = (float) row;
                rowptr[col] = value;
            }
        }
    }

    cuda(MallocArray( &texArray, 
                                  &channelDesc, 
                                  inWidth, 
                                  inHeight));

    cuda(Memcpy2DToArray( texArray, 0, 0, 
                                      texContents, inWidth*sizeof(T), 
                                      inWidth*sizeof(T), 
                                      inHeight, 
                                      hipMemcpyHostToDevice));
    cuda(BindTextureToArray(tex, texArray));

    outPitch = outWidth*sizeof(float4);
    outPitch = (outPitch+0x3f)&~0x3f;

    cuda(HostAlloc( (void **) &outHost, outWidth*outPitch, hipHostMallocMapped));
    cuda(HostGetDevicePointer( (void **) &outDevice, outHost, 0 ));

    tex.filterMode = filterMode;
    tex.addressMode[0] = addressModeX;
    tex.addressMode[1] = addressModeY;
    blocks.x = 2;
    blocks.y = 1;
    threads.x = 64; threads.y = 4;
    TexReadout<<<blocks,threads>>>( outDevice, outWidth, outPitch, outHeight, base, increment );
    cuda(DeviceSynchronize());

    for ( int row = 0; row < outHeight; row++ ) {
        float4 *outrow = (float4 *) ((char *) outHost + row*outPitch);
        for ( int col = 0; col < outWidth; col++ ) {
            printf( "(%.1f, %.1f) ", outrow[col].z, outrow[col].w );
        }
        printf( "\n" );
    }
    printf( "\n" );

Error:
    if ( ! initTex ) free( texContents );
    hipFreeArray( texArray );
    hipHostFree( outHost );
}

int
main( int argc, char *argv[] )
{
    int ret = 1;
    hipError_t status;

    cuda(SetDeviceFlags(hipDeviceMapHost));
    cuda(Free(0));

    // go through once each with linear and point filtering
    do {
        tex.normalized = false;
        tex.filterMode = hipFilterModePoint;
        tex.addressMode[0] = hipAddressModeClamp;
        tex.addressMode[1] = hipAddressModeClamp;

        float2 base, increment;
        base.x = 0.0f;//-1.0f;
        base.y = 0.0f;//-1.0f;
        increment.x = 1.0f;
        increment.y = 1.0f;
//        CreateAndPrintTex<float2>( NULL, 8, 8, 8, 8, base, increment, tex.filterMode, tex.addressMode[0], tex.addressMode[1] );

        CreateAndPrintTex<float2>( NULL, 256, 256, 256, 256, base, increment, tex.filterMode, tex.addressMode[0], tex.addressMode[1] );


    } while ( tex.filterMode == hipFilterModeLinear );

    ret = 0;
Error:
    return ret;
}
