#include "hip/hip_runtime.h"
/*
 *
 * tex1dfetch_int2float.cu
 *
 * Microdemo for the method used by GPU texturing hardware to
 * promote integers to unitized floats.
 *
 * Build with: nvcc -I ../chLib <options> tex1dfetch_int2float.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <assert.h>

#include <chError.h>

texture<signed char, 1, hipReadModeNormalizedFloat> tex;

extern "C" __global__ void
TexReadout( float *out, size_t N )
{
    for ( size_t i = blockIdx.x*blockDim.x + threadIdx.x; 
          i < N; 
          i += gridDim.x*blockDim.x )
    {
        out[i] = tex1Dfetch( tex, i );
    }
}

template<class T> float TexPromoteToFloat( T ) { return 0.0f; }

float
TexPromoteToFloat( signed char c )
{
    if ( c == (signed char) 0x80 ) {
        return -1.0f;
    }
    return (float) c / 127.0f;
}

float
TexPromoteToFloat( short s )
{
    if ( s == (short) 0x8000 ) {
        return -1.0f;
    }
    return (float) s / 32767.0f;
}

float
TexPromoteToFloat( unsigned char uc )
{
    return (float) uc / 255.0f;
}

float
TexPromoteToFloat( unsigned short us )
{
    return (float) us / 65535.0f;
}

template<class T>
void
CheckTexPromoteToFloat( size_t N )
{
    T *inHost, *inDevice;
    float *foutHost, *foutDevice;
    hipError_t status;

    cuda(HostAlloc( (void **) &inHost, 
                                N*sizeof(T), 
                                hipHostMallocMapped));
    cuda(HostGetDevicePointer( (void **) &inDevice, 
                                           inHost, 
                                           0 ));
    cuda(HostAlloc( (void **) &foutHost, 
                                N*sizeof(float), 
                                hipHostMallocMapped));
    cuda(HostGetDevicePointer( (void **) &foutDevice, 
                                           foutHost, 
                                           0 ));

    for ( int i = 0; i < N; i++ ) {
        inHost[i] = (T) i;
    }
    memset( foutHost, 0, N*sizeof(float) );

    cuda(BindTexture( NULL, 
                      tex, 
                      inDevice, 
                      hipCreateChannelDesc<T>(), 
                      N*sizeof(T)));
    TexReadout<<<2,384>>>( foutDevice, N );
    cuda(DeviceSynchronize());

    for ( int i = 0; i < N; i++ ) {
        printf( "%.2f ", foutHost[i] );
        assert( foutHost[i] == TexPromoteToFloat( (T) i ) );
    }
    printf( "\n" );
Error:
    hipHostFree( inHost );
    hipHostFree( foutHost );
}

int
main( int argc, char *argv[] )
{
    int ret = 1;
    hipError_t status;

    cuda(SetDeviceFlags(hipDeviceMapHost));
    cuda(Free(0));
    CheckTexPromoteToFloat<signed char>( 256 );
    CheckTexPromoteToFloat<unsigned char>( 256 );

    CheckTexPromoteToFloat<short>( 65536 );
    CheckTexPromoteToFloat<unsigned short>( 65536 );

    ret = 0;
Error:
    return ret;
}
