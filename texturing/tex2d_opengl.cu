#include "hip/hip_runtime.h"
/*
 *
 * tex2d_opengl.cu
 *
 * Microdemo to illustrate the workings of 2D texturing.
 *
 * Build with: nvcc -I ../chLib <options> tex2d_opengl.cu, with
 * platform-specific OpenGL includes and libs.
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>

#include <chError.h>

#include <GL/glut.h>

int g_width, g_height;
float g_scale = 1.0f;
char *g_texture;

hipArray *g_arrayTexture;
uchar4 *g_hostFrameBuffer;
uchar4 *g_deviceFrameBuffer;
bool g_displayText = true;

char *
LoadTexturePPM( const char * filename )
{
    int width, height, maxval;
    FILE * file = NULL;
    char *data = NULL;

    // open texture data
    file = fopen( filename, "rb" );
    if ( file == NULL )
        goto Error;

    if ( 'P' != fgetc( file ) )
        goto Error;
    if ( '6' != fgetc( file ) )
        goto Error;
    if ( 1 != fscanf( file, "%d", &width ) )
        goto Error;
    if ( 1 != fscanf( file, "%d", &height ) )
        goto Error;
    if ( 1 != fscanf( file, "%d", &maxval ) )
        goto Error;
    if ( maxval != 0xff )
        goto Error;

    {
        int ch;
        do {
            ch = fgetc( file );
        } while ( isspace( ch ) );
    }

    // allocate buffer
    data = (char *) malloc( width * height * 3 );
    if ( ! data )
        goto Error;

    // read texture data
    if ( 1 != fread( data, width * height * 3, 1, file ) )
        goto Error;

    fclose( file );
    return data;
Error:
    if ( file ) {
        fclose( file );
    }
    free( data );
    return NULL;
}

texture<uchar4, 2, hipReadModeElementType> tex2d;

hipError_t
CreateAndPopulateArray( hipArray **ret, char *base, int width, int height )
{
    uchar4 *array4 = new uchar4[width*height];
    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
    hipError_t status = hipMallocArray( ret, &desc, width, height );
    if ( hipSuccess != status ) {
        return status;
    }
    for ( int row = 0; row < 256; row++ ) {
        char *baserow = base+(height-row-1)*3*width;
        for ( int col = 0; col < 256; col++ ) {
            uchar4 value;
            value.z = *baserow++;
            value.y = *baserow++;
            value.x = *baserow++;
            value.w = 0;
            array4[row*width+col] = value;
        }
    }

    return hipMemcpy2DToArray( *ret, 0, 0, array4, width*4, width*4, height, hipMemcpyHostToDevice );
}

__global__ void
RenderTextureUnnormalized( uchar4 *out, int width, int height )
{
    for ( int j = blockIdx.x; j < height; j += gridDim.x ) {
        int row = height-j-1;
        for ( int col = threadIdx.x; col < width; col += blockDim.x ) {
            out[row*width+col] = tex2D( tex2d, (float) col, (float) row );
        }
    }
}

__global__ void
RenderTextureNormalized( uchar4 *out, 
                         int width, 
                         int height, 
                         int scale )
{
    for ( int j = blockIdx.x; j < height; j += gridDim.x ) {
        int row = height-j-1;
        out = (uchar4 *) (((char *) out)+row*4*width);
        float texRow = scale * (float) row / (float) height;
        float invWidth = scale / (float) width;
        for ( int col = threadIdx.x; col < width; col += blockDim.x ) {
            float texCol = col * invWidth;
            out[col] = tex2D( tex2d, texCol, texRow );
        }
    }
}

void glPrint(int x, int y, const char *s, void *font)
{
    glColor3f(1.0, 1.0, 1.0);
    
    glRasterPos2f(x, y);
    int len = (int) strlen(s);
    for (int i = 0; i < len; i++) {
        glutBitmapCharacter(font, s[i]);
    }
}

int g_idxAddressMode;

void displayCB(void)		/* function called whenever redisplay needed */
{
    glClear(GL_COLOR_BUFFER_BIT);

    if ( hipSuccess != hipBindTextureToArray( tex2d, 
                                                g_arrayTexture, 
                                                hipCreateChannelDesc<uchar4>() ) )
    {
        return;
    }
    if ( tex2d.normalized ) {
        RenderTextureNormalized<<<g_height, 384>>>( g_deviceFrameBuffer, g_width, g_height, g_scale );
    }
    else {
        RenderTextureUnnormalized<<<g_height, 384>>>( g_deviceFrameBuffer, g_width, g_height );
    }
    if ( hipSuccess != hipDeviceSynchronize() )
        return;
    glRasterPos2f( 0.0f, 0.0f );
    glDrawPixels( g_width, g_height, GL_RGBA, GL_UNSIGNED_BYTE, g_hostFrameBuffer );
    if ( g_displayText ) {
        char s[256];
        int fontWidth = 9;      // width of 9x15 font
        int x = fontWidth;      // one character of 9x15 font
        int fontHeight = 15;

        int y = g_height - fontHeight;
        if ( tex2d.normalized ) {
            glPrint( x, y, "Normalized (hit N for unnormalized)", GLUT_BITMAP_9_BY_15 );
        }
        else {
            glPrint( x, y, "Unnormalized (hit N for normalized)", GLUT_BITMAP_9_BY_15 );
        }
        y -= fontHeight;
        switch ( tex2d.addressMode[0] ) {
            case hipAddressModeClamp:
                sprintf( s, "X address mode: Clamp %s", g_idxAddressMode ? "\0" :
                    "(W=wrap, M=mirror, B=border)" );
                break;
            case hipAddressModeWrap:
                sprintf( s, "X address mode: Wrap %s", g_idxAddressMode ? "\0" :
                    "(C=clamp, M=mirror, B=border)" );
                break;
            case hipAddressModeMirror:
                sprintf( s, "X address mode: Mirror %s", g_idxAddressMode ? "\0" :
                    "(C=clamp, W=wrap, B=border)" );
                break;
            case hipAddressModeBorder:
                sprintf( s, "X address mode: Border %s", g_idxAddressMode ? "\0" :
                    "(C=clamp, W=wrap, M=mirror)" );
                break;
        }
        glPrint( x, y, s, GLUT_BITMAP_9_BY_15 );
        y -= fontHeight;

        switch ( tex2d.addressMode[1] ) {
            case hipAddressModeClamp:
                sprintf( s, "Y address mode: Clamp %s", !g_idxAddressMode ? "\0" :
                    "(W=wrap, M=mirror, B=border)" );
                break;
            case hipAddressModeWrap:
                sprintf( s, "Y address mode: Wrap %s", !g_idxAddressMode ? "\0" :
                    "(C=clamp, M=mirror, B=border)" );
                break;
            case hipAddressModeMirror:
                sprintf( s, "Y address mode: Mirror %s", !g_idxAddressMode ? "\0" :
                    "(C=clamp, W=wrap, B=border)" );
                break;
            case hipAddressModeBorder:
                sprintf( s, "Y address mode: Border %s", !g_idxAddressMode ? "\0" :
                    "(C=clamp, W=wrap, M=mirror)" );
                break;
        }
        glPrint( x, y, s, GLUT_BITMAP_9_BY_15 );
        y -= fontHeight;
        glPrint( x, y, "Hit X to set X addressing mode, Y to set Y addressing mode", GLUT_BITMAP_9_BY_15 );
        y -= fontHeight;
        glPrint( x, y, "When in normalized mode, hit 1-9 keys to set scale", GLUT_BITMAP_9_BY_15 );
        y -= fontHeight;
        glPrint( x, y, "T key toggles text display", GLUT_BITMAP_9_BY_15 );
    }
    glFlush();				/* Complete any pending operations */
}

void keyCB(unsigned char key, int x, int y)	/* called on key press */
{
    switch ( key ) {
        case '1': case '2': case '3':
        case '4': case '5': case '6':
        case '7': case '8': case '9':
            g_scale = (float) (key-'0');
            break;
        case 'q':
            exit(0);
            break;
        case 'x':
            g_idxAddressMode = 0;
            break;
        case 'y':
            g_idxAddressMode = 1;
            break;
        case 'n':
            tex2d.normalized = ! tex2d.normalized;
            break;
        case 'w':
            tex2d.addressMode[g_idxAddressMode] = hipAddressModeWrap;
            break;
        case 'c':
            tex2d.addressMode[g_idxAddressMode] = hipAddressModeClamp;
            break;
        case 'm':
            tex2d.addressMode[g_idxAddressMode] = hipAddressModeMirror;
            break;
        case 'b':
            tex2d.addressMode[g_idxAddressMode] = hipAddressModeBorder;
            break;
        case 't':
            g_displayText = ! g_displayText;
            break;
        default: 
            return;
    }
    glutPostRedisplay();
}

void
reshapeCB( int width, int height )
{
    hipError_t status;

    if ( g_hostFrameBuffer ) {
        hipHostFree( g_hostFrameBuffer );
    }

    g_width = width;
    g_height = height;

    status = hipHostAlloc( &g_hostFrameBuffer, g_width*sizeof(uchar4)*g_height, hipHostMallocMapped );
    if ( hipSuccess != status ) {
        goto Error;
    }
    status = hipHostGetDevicePointer( &g_deviceFrameBuffer, g_hostFrameBuffer, 0 );
    if ( hipSuccess != status ) {
        goto Error;
    }
    glMatrixMode( GL_PROJECTION );
    glLoadIdentity();
    gluOrtho2D(0,width,0,height);
    glViewport(0,0,width,height);
    glutPostRedisplay();
Error:;
}

int
main(int argc, char *argv[])
{
    hipError_t status;
    int ret = 1;

    g_width = 512;
    g_height = 512;

    glutInit(&argc, argv);

    glutInitDisplayMode(GLUT_RGB);
    glutInitWindowSize(g_width,g_height);
    glutCreateWindow("CUDA 2D Texturing");

    g_texture = LoadTexturePPM( "TextureDemoImage.ppm" );
    if ( ! g_texture ) {
        fprintf( stderr, "Could not load texture\n");
        goto Error;
    }

    cuda(SetDeviceFlags( hipDeviceMapHost ) );

    CUDART_CHECK( CreateAndPopulateArray( &g_arrayTexture, g_texture, 256, 256 ) );

    glClearColor(0.0,0.0,0.0,0.0);
    glutDisplayFunc(displayCB);
    glutKeyboardFunc(keyCB);
    glutReshapeFunc(reshapeCB);

    glutMainLoop();

    // we never get here

    ret = 0;
Error:
    return ret;
}
