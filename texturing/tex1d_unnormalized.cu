#include "hip/hip_runtime.h"
/*
 *
 * tex1d_unnormalized.cu
 *
 * Microdemo to illustrate how to texture using unnormalized
 * texture coordinates in the range [0..Dim), not [0..1).
 *
 * Build with: nvcc -I ../chLib <options> tex1dfetch_unnormalized.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include <chError.h>

texture<float, 1> tex;

extern "C" __global__ void
TexReadout( float2 *out, size_t N, float base, float increment )
{
    for ( size_t i = blockIdx.x*blockDim.x + threadIdx.x; 
          i < N; 
          i += gridDim.x*blockDim.x )
    {
        float x = base + (float) i * increment;
        out[i].x = x;
        out[i].y = tex1D( tex, x );
    }
}

/*
#ifdef PRINT_HEXADECIMAL_FLOATS
        printf( "(%.2f, 0x%08x)\n", outHost[i].x, *(int *) (&outHost[i].y) );
#else
        printf( "(%.2f, %.2f)\n", outHost[i].x, outHost[i].y );
#endif
*/

template<class T>
void
CreateAndPrintTex( T *initTex, size_t texN, size_t outN, 
    float base, float increment, 
    hipTextureFilterMode filterMode = hipFilterModePoint, 
    hipTextureAddressMode addressMode = hipAddressModeClamp )
{
    T *texContents = 0;
    hipArray *texArray = 0;

    float2 *outHost = 0, *outDevice = 0;
    hipError_t status;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();

    // use caller-provided array, if any, to initialize texture
    if ( initTex ) {
        texContents = initTex;
    }
    else {
        // default is to initialize with identity elements
        texContents = (T *) malloc( texN*sizeof(T) );
        if ( ! texContents )
            goto Error;
        for ( int i = 0; i < texN; i++ ) {
            texContents[i] = (T) i;
        }
    }

    cuda(MallocArray(&texArray, &channelDesc, texN));

    cuda(HostAlloc( (void **) &outHost, 
                                outN*sizeof(float2), 
                                hipHostMallocMapped));
    cuda(HostGetDevicePointer( (void **) 
                                           &outDevice, 
                                           outHost, 0 ));

    cuda(MemcpyToArray( texArray, 
                                    0, 0, 
                                    texContents, 
                                    texN*sizeof(T), 
                                    hipMemcpyHostToDevice));
    cuda(BindTextureToArray(tex, texArray));

    tex.filterMode = filterMode;
    tex.addressMode[0] = addressMode;
    cuda(HostGetDevicePointer(&outDevice, outHost, 0));
    TexReadout<<<2,384>>>( outDevice, outN, base, increment );
    cuda(DeviceSynchronize());

    for ( int i = 0; i < outN; i++ ) {
        printf( "(%.2f, %.2f)\n", outHost[i].x, outHost[i].y );
    }
    printf( "\n" );

Error:
    if ( ! initTex ) free( texContents );
    if ( texArray ) hipFreeArray( texArray );
    if ( outHost ) hipHostFree( outHost );
}

int
main( int argc, char *argv[] )
{
    int ret = 1;
    hipError_t status;
    cuda(SetDeviceFlags(hipDeviceMapHost));

    //CreateAndPrintTex<float>( NULL, 8, 8, 0.0f, 1.0f, hipFilterModePoint );
    //CreateAndPrintTex<float>( NULL, 8, 8, 0.0f, 1.0f, hipFilterModeLinear );
//    CreateAndPrintTex<float>( NULL, 8, 20, 0.9f, 0.01f, hipFilterModePoint );
    {
        float texData[10];
        for ( int i = 0; i < 10; i++ ) {
            texData[i] = (float) i / 10.0f;
        }
//        CreateAndPrintTex<float>( texData, 10, 10, 0.0f, 1.0f, hipFilterModePoint );
        CreateAndPrintTex<float>( texData, 10, 10, 1.5f, 0.1f, hipFilterModeLinear );
    }
    ret = 0;
Error:
    return ret;
}
