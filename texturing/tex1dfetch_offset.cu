#include "hip/hip_runtime.h"
/*
 *
 * tex1dfetch_offset.cu
 *
 * Microdemo for the offset passback parameter when binding
 * a texture to device memory.
 *
 * Build with: nvcc -I ../chLib <options> tex1dfetch_offset.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include <chError.h>

#define NUM_FLOATS 4096

texture<float, 1, hipReadModeElementType> tex;

__global__ void
TexReadout( float *out, size_t offset, size_t N )
{
    for ( size_t i = blockIdx.x*blockDim.x + threadIdx.x; 
          i < N; 
          i += gridDim.x*blockDim.x )
    {
        out[i] = tex1Dfetch( tex, i + (int) offset );
    }
}

bool
CheckTex( float *hostOut, const float *in, size_t offset, size_t N )
{
    float *deviceOut;
    hipError_t status;
    bool ret = false;
    memset( hostOut, 0, N*sizeof(float) );
    cuda(HostGetDevicePointer( (void **) &deviceOut, hostOut, 0 ));
    
    TexReadout<<<2,384>>>( deviceOut, offset>>2, N );
    cuda(DeviceSynchronize());
    for ( int i = 0; i < N; i++ ) {
        if ( in[i] != hostOut[i] ) {
            printf( "Mismatch at index %d\n", i );
            goto Error;
        }
    }
    ret = true;
Error:
    return ret;
}

int
main( int argc, char *argv[] )
{
    int ret = 1;
    float *deviceTex = 0;
    float fInit[NUM_FLOATS];

    float *foutHost = 0;
    float *foutDevice = 0;
    hipError_t status;
    hipDeviceProp_t props;
    size_t offset;

    cuda(SetDeviceFlags(hipDeviceMapHost));
    cuda(GetDeviceProperties( &props, 0));
    printf( "Base texture alignment requirement: %d bytes\n", (int) props.textureAlignment );

    for ( int i = 0; i < NUM_FLOATS; i++ ) {
        fInit[i] = (float) i;
    }

    cuda(Malloc( (void **) &deviceTex, 2*NUM_FLOATS*sizeof(float)) );
    cuda(HostAlloc( (void **) &foutHost, NUM_FLOATS*sizeof(float), hipHostMallocMapped));
    cuda(HostGetDevicePointer( (void **) &foutDevice, foutHost, 0 ));

    for ( offset = 0; offset <= NUM_FLOATS/2; offset += 4 )
    {
        size_t texOffset;
        cuda(Memset(deviceTex, 0xcc, 2*NUM_FLOATS*sizeof(float)));
        cuda(Memcpy(deviceTex+offset, fInit, NUM_FLOATS*sizeof(float), hipMemcpyHostToDevice));

        cuda(BindTexture( &texOffset, tex, deviceTex+offset, NUM_FLOATS*sizeof(float)) );
        printf( "My offset = %d, texture offset = %d\n", (int) offset, (int) texOffset );

        if ( ! CheckTex( foutHost, fInit, texOffset, NUM_FLOATS ) ) {
            goto Error;
        }
    }
    ret = 0;
Error:
    hipHostFree( foutHost );
    hipFree( deviceTex );
    return ret;
}
